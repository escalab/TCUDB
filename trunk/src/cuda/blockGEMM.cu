#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include "blockGEMM.h"

void PrintMatrix(char name[], int rows, int cols, const float* m){
  printf("%s\n", name);
  for(int row = 0; row < rows; ++row){
	for(int col = 0; col < cols; ++col){
		printf("%f ", m[row * cols + col]);
	}
	printf("\n");
  }
}

void copyElements(float* out, float* entry, unsigned long long eRows, unsigned long long eCols, unsigned long long oRows, unsigned long long oCols, unsigned long long x, unsigned long long y,
	unsigned long long ofA, unsigned long long ofB, float beta){
	unsigned long long counterRows = eRows;
	unsigned long long counterCols = eCols;
	if(ofA){
		counterRows = ofA;
	}
	if(ofB){
		counterCols = ofB;	
	}
	for(unsigned long long i = 0; i < counterRows; ++i){
		for(unsigned long long j = 0; j < counterCols; ++j){
			unsigned long long index = x*eRows*oCols + (i*oCols) + (y*eCols + j);
			out[index] = entry[i*eCols + j] + beta * out[index];
		}

	}
}

float * doMultiply2Matrices(
        int a1Rows, int a1Cols,  float * A1,
        int a2Rows, int a2Cols,  float * A2,
	float* C, float alpha)
{
    float beta = 0;
    hipblasHandle_t  handle;

    hipblasCreate (&handle) ;

    hipblasSgemm(handle,HIPBLAS_OP_N, HIPBLAS_OP_N,
                  a2Cols, a1Rows, a1Cols,
                  &alpha,
                  A2, a2Cols,
                  A1, a1Cols,
                  &beta,
                  C, a2Cols );

    hipblasDestroy ( handle ) ;

    return C ;


}

float * doSplitStreaming(
        int a1Rows, int a1Cols, half * A1,
        int a2Rows, int a2Cols, half * A2,
        float * C, hipStream_t cudaStream, hipblasHandle_t handle)
{
    float alpha = 1.0f, beta = 0.0f;

    hipblasSetStream(handle, cudaStream);

    hipblasStatus_t stat = hipblasGemmEx(handle,
                  HIPBLAS_OP_N, HIPBLAS_OP_N,
                  a2Cols, a1Rows, a1Cols,
                  &alpha,
                  A2, HIP_R_16F, a2Cols,
                  A1, HIP_R_16F, a1Cols,
                  &beta,
                  C, HIP_R_32F, a2Cols,
                  //HIPBLAS_COMPUTE_32F_PEDANTIC, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
                  HIPBLAS_COMPUTE_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    printf("cublas status = %d\n", stat);

    return C ;
}

float * doMultiply2MatricesStreaming(
        int a1Rows, int a1Cols,  float * A1,
        int a2Rows, int a2Cols,  float * A2,
	float* C, hipStream_t cudaStream, hipblasHandle_t handle, float alpha)
{

    float beta =  0.0;

    hipblasSetStream(handle, cudaStream) ;

    hipblasStatus_t stat = hipblasSgemm(handle,HIPBLAS_OP_N, HIPBLAS_OP_N,
                  a2Cols, a1Rows, a1Cols,
                  &alpha,
                  A2, a2Cols,
                  A1, a1Cols,
                  &beta,
                  C, a2Cols );
    printf("cublas status = %d\n", stat);

    return C ;
}

