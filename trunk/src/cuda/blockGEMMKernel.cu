#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <pthread.h>
#include <unistd.h>
#include "../include/common.h"
#include "../include/blockJoin.h"
#include "blockGEMM.h"

const int num_submatrix = 4;
const int numStreams = 2;
const int num_threads = numStreams;

struct thread_args{
	int threadId;
	unsigned long long overflowA;
	unsigned long long numSubMatrixA;
	unsigned long long subRows;
	unsigned long long subCols;
	unsigned long long m;
	unsigned long long n;
	unsigned long long k;
	unsigned long long y;
	unsigned long long i;
	float *C;
	float *A;

};

volatile int running_threads = 0;
pthread_mutex_t running_mutex = PTHREAD_MUTEX_INITIALIZER;


hipStream_t streams[numStreams];
half* b = 0;
half* a[num_threads];
half* a_h[num_threads];

float* c[num_threads];
float* c_h[num_threads];
struct thread_args targs[num_threads];
pthread_t threads[num_threads];
char threads_active[num_threads];
hipblasHandle_t handles[num_threads];

void blockGEMM_half(char transa, char transb,
        unsigned long long m, unsigned long long n, unsigned long long k,
        float alpha,
        half *A, int lda,
        half *B, int ldb,
        float beta,
        float *C, int ldc)
{
    
    printf("blockGEMM_half\n");
    half* A_d;
    half* B_d;
    float* C_d;
    unsigned long long A_sz = m * k;
    unsigned long long B_sz = n * k;
    unsigned long long C_sz = m * n;
    unsigned long long MAX =  (unsigned long long )m* (unsigned long long) n / num_submatrix;
    
	MAX -= MAX % k;
	printf("MAX: %d\n", MAX);
	printf("B_sz: %d\n",B_sz);
	unsigned long long numSubMatrixB = B_sz / MAX;
	printf("SubmatriciesB: %d\n", numSubMatrixB);
	unsigned long long SMB_sz = B_sz / numSubMatrixB;
	printf("SMB_sz: %d\n", SMB_sz);
	unsigned long long subCols = B_sz / (numSubMatrixB * k);
	printf("subCols: %d\n", subCols);
	
	unsigned long long numSubMatrixA = A_sz / MAX;
	unsigned long long SMA_sz = A_sz / numSubMatrixA;
	unsigned long long subRows = A_sz / (numSubMatrixA * k);
	printf("subrows: %d\n", subRows);
	printf("SMA_sz: %d\n", SMA_sz);
	printf("submatriciesA: %d\n", numSubMatrixA);
	unsigned long long overflowA = m % subRows;
	unsigned long long overflowB = n % subCols;
	printf("overflowB: %d\n", overflowB);
	printf("overflowA: %d\n", overflowA);
	hipMalloc((void**) &b, sizeof(half) * subCols * k);
	for(int i = 0; i < numStreams; ++i){
		hipblasCreate(&handles[i]);
		hipStreamCreate(&streams[i]);
		hipMalloc((void**) &a[i], sizeof(half) * subRows * k);
		hipMalloc((void**) &c[i], sizeof(float) * subCols * subRows);
        // allocate pinned memory on the CPU
		hipHostMalloc((void**) &a_h[i], sizeof(half) * subRows * k);
		hipHostMalloc((void**) &c_h[i], sizeof(float) * subCols * subRows);
		threads_active[i] = 0;
	}

	half* temp3 = 0;
	
	hipHostMalloc((void**) &temp3, sizeof(half)*subCols * k );
	for(unsigned long long i = 0; i < numSubMatrixB + 1; ++i){
		int count = 0;
		if(overflowB == 0 && i == numSubMatrixB){
			break;
		}
	
		for(int j = 0; j < k; ++j){
			for(int x = 0; x < subCols; ++x){
				if(i * subCols + x < n){
					temp3[j * subCols + x] = B[j * n + (i*subCols + x)];
				}else{
					temp3[j *subCols + x] = 0;
				}
			}
		}
	
		hipMemcpyAsync(b, temp3, sizeof(half)*subCols*k, hipMemcpyHostToDevice, streams[0]);
		unsigned long long y = 0;
		int streamsActive = 0;
		while(y < numSubMatrixA + 1){
			if(overflowA == 0 && y == numSubMatrixA){
				break;
			}
			for(int j = 0; j < subRows; ++j){
				for(int x = 0; x < k; ++x){
					if(y * subRows + j < m){
						(a_h[y % numStreams])[j * k + x] = A[y*subRows*k + j*k + x];
					}else{
						(a_h[y % numStreams])[j * k + x] = 0;
					}
				}			
			}
			
			hipMemcpyAsync(a[y % numStreams], a_h[y % numStreams], sizeof(half)*subRows*k, hipMemcpyHostToDevice, streams[y % numStreams]);
			printf("sending multiply %d,%d to stream %d\n", y, i, y % numStreams);
            printf("perform hipblasGemmEx with FP16 inputs\n");
            // FIXME: which function?
            doMMStreaming(subRows, k, a[y % numStreams], k,
                subCols, b, c[y % numStreams], streams[y % numStreams],
                handles[y % numStreams]);
            
            // copy result back to the host
			hipMemcpyAsync(c_h[y % numStreams], c[y % numStreams], sizeof(float)*subRows*subCols, hipMemcpyDeviceToHost, streams[y % numStreams]);
						
			streamsActive++;
			if(y % numStreams == numStreams - 1){
				for(int s = 0; s < numStreams; ++s){
					hipStreamSynchronize(streams[s]);
					int currWork = count * numStreams + s;
					if(i == numSubMatrixB && currWork == numSubMatrixA){
						copyElements(C,  c_h[s], subRows, subCols, m, n, currWork, i, overflowA, overflowB, beta);
					}else if(i == numSubMatrixB){
						copyElements(C,  c_h[s], subRows, subCols, m, n, currWork, i, 0, overflowB, beta);
					}else if(currWork == numSubMatrixA){
						copyElements(C,  c_h[s], subRows, subCols, m, n, currWork, i, overflowA, 0, beta);
					}else{
						copyElements(C, c_h[s], subRows, subCols, m, n, currWork, i, 0, 0, beta);
					}
					streamsActive--;
				}
				++count;
			}
			++y;

		}
		//PrintMatrix("C", m, n, C);
		printf("%d Streams Active Left over\n", streamsActive);
		for(int s = 0; s < streamsActive; ++s){
			hipStreamSynchronize(streams[s]);
			int currWork = count * numStreams + s;
			if(i == numSubMatrixB && currWork == numSubMatrixA){
				copyElements(C,  c_h[s], subRows, subCols, m, n, currWork, i, overflowA, overflowB, beta);
			}else if(i == numSubMatrixB){
				copyElements(C,  c_h[s], subRows, subCols, m, n, currWork, i, 0, overflowB, beta);
			}else if(currWork == numSubMatrixA){
				copyElements(C,  c_h[s], subRows, subCols, m, n, currWork, i, overflowA, 0, beta);
			}else{
				copyElements(C, c_h[s], subRows, subCols, m, n, currWork, i, 0, 0, beta);
			}

		}
		
		
		
	
	}

	for(int i = 0; i < numStreams; ++i){
		hipFree(a[i]);
		hipFree(c[i]);
		hipHostFree(a_h[i]);
		hipHostFree(c_h[i]);
		hipStreamDestroy(streams[i]);
	}
	hipFree(b);
	hipHostFree(temp3);
    
}

struct tableNode * blockJoin(struct joinNode *jNode, struct statistic *pp)
{
    struct tableNode * res = NULL;

    


    return res;
}

