#include "hip/hip_runtime.h"
/*
   Copyright (c) 2012-2013 The Ohio State University.

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.
*/


#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string.h>
#include <time.h>
#include "../include/common.h"
#include "../include/gpuCudaLib.h"
#include "../include/cudaHash.h"
#include "scanImpl.cu"
#include "../include/cuPrintf.cu"
#include "../include/cuPrintf.cuh"

/*
#define ERROR_CHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}
*/

/*
 * Combine the group by columns to build the group by keys. 
 */

__global__ static void build_groupby_key(char ** content, int gbColNum, int * gbIndex, int * gbType, int * gbSize, long tupleNum, int * key, int *num, int* groupNum){

    int stride = blockDim.x * gridDim.x;
    int offset = blockIdx.x * blockDim.x + threadIdx.x;

    for(long i = offset; i< tupleNum; i+= stride){
        char buf[128] = {0};
        for (int j=0; j< gbColNum; j++){
            char tbuf[32]={0};
            int index = gbIndex[j];

            if (index == -1){
                gpuItoa(1,tbuf,10);
                gpuStrncat(buf,tbuf,1);

            }else if (gbType[j] == STRING){
                gpuStrncat(buf, content[index] + i*gbSize[j], gbSize[j]);

            }else if (gbType[j] == INT){
                int key = ((int *)(content[index]))[i];
                gpuItoa(key,tbuf,10);
                gpuStrcat(buf,tbuf);
            }
        }
        int hkey = StringHash(buf) % HSIZE;
        key[i]= hkey;
        num[hkey] = 1; // gb count +1
        atomicAdd(&(groupNum[hkey]), 1);
    }
}


/*
 * Count the number of groups 
 */

__global__ static void count_group_num(int *num, int tupleNum, int *totalCount){
        int stride = blockDim.x * gridDim.x;
        int offset = blockIdx.x * blockDim.x + threadIdx.x;
        int localCount = 0;

        for(int i=offset; i<tupleNum; i+= stride){
                if(num[i] == 1){
                        localCount ++;
                }
        }

        atomicAdd(totalCount,localCount);
}

/*
 * Calculate the groupBy expression.
 */

__device__ static float calMathExp(char **content, struct mathExp exp, int pos){
    float res;

    // terminate condition
    if(exp.op == NOOP){
        // opType -- regular column or a constant
        if (exp.opType == CONS)
            res = exp.opValue;
        else{
            int index = exp.opValue;
            int type  = exp.dataType;
            //cuPrintf("pos for content: %d\n", pos); // 0-15
            if (type == INT) {
                res = ((int *)(content[index]))[pos];
            } else { // type == FLOAT
                res = ((float *)(content[index]))[pos];
            }
            //res = ((int *)(content[index]))[pos];
            //res = ((float *)(content[index]))[pos];
        }
    
    }else if(exp.op == PLUS ){
        res = calMathExp(content, ((struct mathExp*)exp.exp)[0],pos) + calMathExp(content, ((struct mathExp*)exp.exp)[1], pos);

    }else if (exp.op == MINUS){
        res = calMathExp(content, ((struct mathExp*)exp.exp)[0],pos) - calMathExp(content, ((struct mathExp*)exp.exp)[1], pos);

    }else if (exp.op == MULTIPLY){
        // NOTE: here only perform multiply, so duplicates may happen
        res = calMathExp(content, ((struct mathExp*)exp.exp)[0],pos) * calMathExp(content, ((struct mathExp*)exp.exp)[1], pos);

    }else if (exp.op == DIVIDE){
        float left = calMathExp(content, ((struct mathExp*)exp.exp)[0],pos);
        float right = calMathExp(content, ((struct mathExp*)exp.exp)[1], pos);
        //cuPrintf("left: %f\tright: %.0f\n", left, right);
        res = left / right;
        //res = calMathExp(content, ((struct mathExp*)exp.exp)[0],pos) / calMathExp(content, ((struct mathExp*)exp.exp)[1], pos);
    }

    return res;
}

/*
 * group by constant. Currently only support SUM function.
 */

__global__ static void agg_cal_cons(char ** content, int colNum, struct groupByExp* exp, long tupleNum, char ** result, float *gpuPageRank){

    int stride = blockDim.x * gridDim.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    float buf[32];

    for(int i=0;i<32;i++)
        buf[i] = 0;

    // peform computation on all matched tuples
    //cuPrintf("tupleNum: %d\n", tupleNum); // 16
    for(int i=index;i<tupleNum;i+=stride){
        for(int j=0;j<colNum;j++){ //j is 0
            int func = exp[j].func;
            if (func == SUM){
                //cuPrintf("opNum: %d\n", exp[j].exp.opNum);//2
                float tmpRes = calMathExp(content, exp[j].exp, i);
                //FIXME: for fair comparison, store into output array
                gpuPageRank[i] = tmpRes;
                //cuPrintf("%.8f\n", tmpRes);
                buf[j] += tmpRes;
            }else if (func == AVG){

                float tmpRes = calMathExp(content, exp[j].exp, i)/tupleNum;
                buf[j] += tmpRes;
            }
        }
    }

    // final result
    for(int i=0;i<colNum;i++)
        atomicAdd(&((float *)result[i])[0], buf[i]);
}

/*
 * gropu by
 */

__global__ static void agg_cal(char ** content, int colNum, struct groupByExp* exp, int * gbType, int * gbSize, long tupleNum, int * key, int *psum, int * groupNum, char ** result){

    int stride = blockDim.x * gridDim.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    for(int i=index;i<tupleNum;i+=stride){

        int hKey = key[i];
        int offset = psum[hKey];

        for(int j=0;j<colNum;j++){
            int func = exp[j].func;
            
            if(func ==NOOP){
                int type = exp[j].exp.opType;

                if(type == CONS){
                    int value = exp[j].exp.opValue;
                    ((int *)result[j])[offset] = value;
                }else{
                    int index = exp[j].exp.opValue;
                    int attrSize = gbSize[j];
                    if(attrSize == sizeof(int)) {
                        ((int *)result[j])[offset] = ((int*)content[index])[i];
                        //No-op
                    }     
                    else
                        memcpy(result[j] + offset*attrSize, content[index] + i * attrSize, attrSize);
                }

            }else if (func == SUM ){
                float tmpRes = calMathExp(content, exp[j].exp, i);

                atomicAdd(& ((float *)result[j])[offset], tmpRes);
                //printf("result: %.0f\tj: %d\toffset: %d\n", ((float *)result[j])[offset], j, offset);
            } else if (func == AVG){
                float tmpRes = calMathExp(content, exp[j].exp, i)/groupNum[hKey];
                atomicAdd(& ((float *)result[j])[offset], tmpRes);
            }
        }
    }
}


/* 
 * groupBy: group by the data and calculate. 
 * 
 * Prerequisite:
 *  input data are not compressed
 *
 * Input:
 *  gb: the groupby node which contains the input data and groupby information
 *  pp: records the statistics such as kernel execution time 
 *
 * Return:
 *  a new table node
 */

struct tableNode * groupBy(struct groupByNode * gb, struct statistic * pp){

    cudaPrintfInit();

    struct timespec start,end;
    struct timespec cudaMemcpy_start,cudaMemcpy_end;
    clock_gettime(CLOCK_REALTIME,&start);
    int *gpuGbIndex = NULL, gpuTupleNum, gpuGbColNum;
    int *gpuGbType = NULL, *gpuGbSize = NULL;

    int *gpuGbKey = NULL;
    char ** gpuContent = NULL, **column = NULL;
    float *gpuPageRank;

    /*
     * @gbCount: the number of groups
     * gbConstant: whether group by constant
     */

    int gbCount;
    int gbConstant = 0;

    //printf("factType: %d\n", gb->table->factType);
    //printf("dimType: %d\n", gb->table->dimType);
    struct tableNode *res = (struct tableNode *) malloc(sizeof(struct tableNode));
    CHECK_POINTER(res);
    res->tupleSize = gb->tupleSize;
    res->totalAttr = gb->outputAttrNum;
    res->attrType = (int *) malloc(sizeof(int) * res->totalAttr);
    CHECK_POINTER(res->attrType);
    res->attrSize = (int *) malloc(sizeof(int) * res->totalAttr);
    CHECK_POINTER(res->attrSize);
    res->attrTotalSize = (int *) malloc(sizeof(int) * res->totalAttr);
    CHECK_POINTER(res->attrTotalSize);
    res->dataPos = (int *) malloc(sizeof(int) * res->totalAttr);
    CHECK_POINTER(res->dataPos);
    res->dataFormat = (int *) malloc(sizeof(int) * res->totalAttr);
    CHECK_POINTER(res->dataFormat);
    res->content = (char **) malloc(sizeof(char **) * res->totalAttr);
    CHECK_POINTER(res->content);

    for(int i=0;i<res->totalAttr;i++){
        res->attrType[i] = gb->attrType[i];
        res->attrSize[i] = gb->attrSize[i];
        res->dataFormat[i] = UNCOMPRESSED;
    }
    
    gpuTupleNum = gb->table->tupleNum;
    gpuGbColNum = gb->groupByColNum;
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuPageRank, gpuTupleNum * sizeof(float *)));
    /*
    float * tmp = (float *)malloc(64);
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(tmp,gb->table->content[0],64,hipMemcpyDeviceToHost));
    for (int k = 0; k < 16; k++) {
        printf("tmp content: %f\n", tmp[k]);
    }
    free(tmp);
    */

    //printf("gpuGbColNum: %d\n", gb->groupByColNum);

    // groupByIndex == -1 means query doesn't contain group by keyword
    if(gpuGbColNum == 1 && gb->groupByIndex[0] == -1){
        gbConstant = 1;
    }


    dim3 grid(1024);
    //dim3 grid(512);
    dim3 block(128);
    int blockNum = gb->table->tupleNum / block.x + 1;
    if(blockNum < 1024)
        grid = blockNum;

    int *gpu_hashNum = NULL, *gpu_psum = NULL, *gpuGbCount = NULL, *gpu_groupNum = NULL;

    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuContent, gb->table->totalAttr * sizeof(char *)));
    column = (char **) malloc(sizeof(char *) * gb->table->totalAttr);
    CHECK_POINTER(column);

    // copy table content for group by operation
    for(int i=0;i<gb->table->totalAttr;i++){
        int attrSize = gb->table->attrSize[i];
        if(gb->table->dataPos[i]==MEM){
            CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)& column[i], attrSize * gb->table->tupleNum));
            CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(column[i], gb->table->content[i], attrSize *gb->table->tupleNum, hipMemcpyHostToDevice));

            CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(&gpuContent[i], &column[i], sizeof(char *), hipMemcpyHostToDevice));
        }else{
            CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(&gpuContent[i], &gb->table->content[i], sizeof(char *), hipMemcpyHostToDevice));
        }
    }

    if(gbConstant != 1){ // query has group by keywords, need build_groupby_key, count_group_num and scanImpl to update gbCount

        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuGbType, sizeof(int) * gb->groupByColNum));
        CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpuGbType,gb->groupByType, sizeof(int) * gb->groupByColNum, hipMemcpyHostToDevice));
        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuGbSize, sizeof(int) * gb->groupByColNum));
        CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpuGbSize,gb->groupBySize, sizeof(int) * gb->groupByColNum, hipMemcpyHostToDevice));


        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuGbKey, gb->table->tupleNum * sizeof(int)));

        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuGbIndex, sizeof(int) * gb->groupByColNum));
        CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpuGbIndex, gb->groupByIndex,sizeof(int) * gb->groupByColNum, hipMemcpyHostToDevice));

        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpu_hashNum,sizeof(int)*HSIZE));
        CUDA_SAFE_CALL_NO_SYNC(hipMemset(gpu_hashNum,0,sizeof(int)*HSIZE));

        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpu_groupNum,sizeof(int)*HSIZE));
        CUDA_SAFE_CALL_NO_SYNC(hipMemset(gpu_groupNum,0,sizeof(int)*HSIZE));

        build_groupby_key<<<grid,block>>>(gpuContent,gpuGbColNum, gpuGbIndex, gpuGbType,gpuGbSize,gpuTupleNum, gpuGbKey, gpu_hashNum, gpu_groupNum);
        CUDA_SAFE_CALL_NO_SYNC(hipDeviceSynchronize());

        CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuGbType));
        CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuGbSize));
        CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuGbIndex));

        gbCount = 1;

        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuGbCount,sizeof(int)));
        CUDA_SAFE_CALL_NO_SYNC(hipMemset(gpuGbCount, 0, sizeof(int)));

        count_group_num<<<grid,block>>>(gpu_hashNum, HSIZE, gpuGbCount);
        CUDA_SAFE_CALL_NO_SYNC(hipDeviceSynchronize());

        // copy groub by count back to host
        CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(&gbCount, gpuGbCount, sizeof(int), hipMemcpyDeviceToHost));

        CUDA_SAFE_CALL(hipMalloc((void**)&gpu_psum,HSIZE*sizeof(int)));
        scanImpl(gpu_hashNum,HSIZE,gpu_psum,pp);

        CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuGbCount));
        CUDA_SAFE_CALL_NO_SYNC(hipFree(gpu_hashNum));
    }

    if(gbConstant == 1)
        res->tupleNum = 1;
    else // query has group by keyword
        res->tupleNum = gbCount;

    printf("[INFO]Number of groupBy results: %ld\n",res->tupleNum);
    // after this point, computation occurs

    char ** gpuResult = NULL;
    char ** result = NULL;
    
    result = (char **)malloc(sizeof(char*)*res->totalAttr); // host stores data address on GPU
    CHECK_POINTER(result);
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpuResult, sizeof(char *)* res->totalAttr));

    // copy thing to device memory for computation
    for(int i=0; i<res->totalAttr;i++){
        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&result[i], res->tupleNum * res->attrSize[i]));
        CUDA_SAFE_CALL_NO_SYNC(hipMemset(result[i], 0, res->tupleNum * res->attrSize[i]));
        res->content[i] = result[i]; 
        res->dataPos[i] = GPU;
        res->attrTotalSize[i] = res->tupleNum * res->attrSize[i];
        CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(&gpuResult[i], &result[i], sizeof(char *), hipMemcpyHostToDevice));
    }


    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuGbType, sizeof(int)*res->totalAttr));
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpuGbType, res->attrType, sizeof(int)*res->totalAttr, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gpuGbSize, sizeof(int)*res->totalAttr));
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpuGbSize, res->attrSize, sizeof(int)*res->totalAttr, hipMemcpyHostToDevice));

    struct groupByExp *gpuGbExp;

    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpuGbExp, sizeof(struct groupByExp)*res->totalAttr));
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpuGbExp, gb->gbExp, sizeof(struct groupByExp)*res->totalAttr, hipMemcpyHostToDevice));
    for(int i=0;i<res->totalAttr;i++){
        struct mathExp * tmpMath;
        if(gb->gbExp[i].exp.opNum == 2){ // 2 operands
            CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&tmpMath, 2* sizeof(struct mathExp)));
            CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(tmpMath,(struct mathExp*)gb->gbExp[i].exp.exp,2*sizeof(struct mathExp), hipMemcpyHostToDevice));
            CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(&(gpuGbExp[i].exp.exp), &tmpMath, sizeof(struct mathExp *), hipMemcpyHostToDevice));
        }
    }

    gpuGbColNum = res->totalAttr; // not sure why update second times
    //printf("2 gpuGbColNum: %d\n", res->totalAttr);

    //verify gpuContent
    /*
    float * tmp2 = (float *)malloc(64);
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(tmp2,gb->table->content[0],64,hipMemcpyDeviceToHost));
    for (int k = 0; k < 16; k++) {
        printf("tmp2 content: %f\n", tmp2[k]);
    }
    free(tmp2);
    */

    if(gbConstant !=1){ // query has group by keyword
        agg_cal<<<grid,block>>>(gpuContent, gpuGbColNum, gpuGbExp, gpuGbType, gpuGbSize, gpuTupleNum, gpuGbKey, gpu_psum, gpu_groupNum,gpuResult);

        CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuGbKey));
        CUDA_SAFE_CALL_NO_SYNC(hipFree(gpu_psum));
        CUDA_SAFE_CALL_NO_SYNC(hipFree(gpu_groupNum));
    }else // query has no group by keyword
        agg_cal_cons<<<grid,block>>>(gpuContent, gpuGbColNum, gpuGbExp, gpuTupleNum,gpuResult, gpuPageRank);

    // verify result on host
    float *h_pageRank = (float *)malloc(gpuTupleNum*sizeof(float));
    clock_gettime(CLOCK_REALTIME,&cudaMemcpy_start);
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(h_pageRank, gpuPageRank, gpuTupleNum*sizeof(float),hipMemcpyDeviceToHost));
    clock_gettime(CLOCK_REALTIME,&cudaMemcpy_end);
    /*
    for (int k = 0; k < gpuTupleNum; k++) {
        printf("PageRank[%d]: %.10f\n", k, h_pageRank[k]);
    }
    */

    /*
    float *h_res = (float *)malloc(sizeof(float));
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(h_res, result[0], sizeof(float),hipMemcpyDeviceToHost));
    printf("host res: %f\n", h_res[0]);
    */ 

    for(int i=0; i<gb->table->totalAttr;i++){
        if(gb->table->dataPos[i]==MEM)
            CUDA_SAFE_CALL_NO_SYNC(hipFree(column[i]));
    }
    
    free(column);
    CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuContent));
    CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuGbType));
    CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuGbSize));
    CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuGbExp));
    CUDA_SAFE_CALL_NO_SYNC(hipFree(gpuResult));

    clock_gettime(CLOCK_REALTIME,&end);
    double timeE = (end.tv_sec -  start.tv_sec)* BILLION + end.tv_nsec - start.tv_nsec;
    double cudaMemcpytime = (cudaMemcpy_end.tv_sec -  cudaMemcpy_start.tv_sec)* BILLION + cudaMemcpy_end.tv_nsec - cudaMemcpy_start.tv_nsec;
    printf("GroupBy Time: %lf\n", timeE/(1000*1000));
    printf("hipMemcpy Time: %lf\n", cudaMemcpytime/(1000*1000));

    cudaPrintfDisplay(stdout, true);
    cudaPrintfEnd();

    return res;
}
