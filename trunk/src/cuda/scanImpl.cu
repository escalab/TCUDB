#ifndef SCAN_IMPL_CU
#define SCAN_IMPL_CU

#include "scan.cu"
#include "../include/common.h"

static void scanImpl(int *d_input, int rLen, int *d_output, struct statistic * pp)
{
	int len = 2;
	if(rLen < len){
		int *input, *output;
		hipMalloc((void**)&input,len*sizeof(int));
		hipMalloc((void**)&output, len*sizeof(int));
		hipMemset(input, 0, len*sizeof(int));
		hipMemcpy(input, d_input, rLen*sizeof(int), hipMemcpyDeviceToDevice);
		preallocBlockSums(len);
		prescanArray(output, input, len, pp);
		deallocBlockSums();
		hipMemcpy(d_output,output,rLen*sizeof(int),hipMemcpyDeviceToDevice);
		hipFree(input);
		hipFree(output);
		return;
	}else{
		preallocBlockSums(rLen);
		prescanArray(d_output, d_input, rLen, pp);
		deallocBlockSums();
	}
}


#endif

