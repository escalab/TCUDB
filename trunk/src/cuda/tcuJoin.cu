#include "hip/hip_runtime.h"
/*
   Copyright (c) 2012-2013 The Ohio State University.

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.
*/


#include <stdio.h>
#include <stdlib.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <sys/fcntl.h>
#include <sys/mman.h>
#include <unistd.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include "../include/common.h"
#include "../include/tcuJoin.h"
#include "../include/gpuCudaLib.h"
#include "scanImpl.cu"
#include <hip/hip_fp16.h>
#include <hiprand.h>
#include <mma.h>
#include <hipblas.h>
#include <math.h>
//#ifdef DEBUG
//#include "../include/cuPrintf.cu"
//#include "../include/cuPrintf.cuh"
//#endif

using namespace nvcuda;

#define MAX_THREADS 1024 // For NVIDIA Turing Architecture

// Define some error checking macros.
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
    if (stat != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
    }
}

#if defined(CUBLAS) || defined(CUBLAS_HALF)
#define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
    }
}
#endif

void* cublasCreateThread(void *x)
{
    hipblasHandle_t* cublasHandle = (hipblasHandle_t *)x;
    cublasErrCheck(hipblasCreate(cublasHandle));
    cublasErrCheck(hipblasSetMathMode(*cublasHandle, HIPBLAS_TENSOR_OP_MATH));
    return NULL;
}

__global__ static void count_op(float *red_sum, int length) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i > length) return;
    if (red_sum[i] != 0)
        return;
}

__global__ static void gb_count(float *red_sum, int length, int *cnt) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i >= length) return;
    if (red_sum[i] != 0)
        atomicAdd(cnt, 1);

}

/* Fill the actual float value for PageRank calculation. 
   Pagerank.ranking and Outdegree.degree */
__global__ void pagerank(char *columnIdx, char *columnVal, int matWidth, half *mat, size_t tupleNum, int attrTypeSize, int attrType, float pagerank_cons) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < tupleNum) {
        int stripe = i * attrTypeSize;
        int *id    = (int*)&columnIdx[stripe];

        if (attrType == INT) {
            int *val = (int*)&columnVal[stripe];
            mat[i*matWidth + (*id)] = __float2half((float)1/(*val));
            //cuPrintf("mat[%d]\t%d\n", i*matWidth + (*id), *val);
        } else if (attrType == FLOAT) {
            float *val   = (float*)&columnVal[stripe];
            
            mat[i*matWidth + (*id)] = __float2half((*val)*pagerank_cons);
            //cuPrintf("mat[%d]\t%.8f\n", i*matWidth + (*id), *val);
        }
    }
}

/* 
 *  Fill 1.0 on the index of unique value in the matrix;
 *  fill 0.0, otherwise. 
 */
__global__ void static gpu_fill(char *column, int matWidth, half *matA, size_t tupleNum, int attrType) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= tupleNum) return;

    int index = i * attrType;
    //int value = (int)column[index]; // char -> int will lose 3 bytes
    int *value   = (int*)&column[index];
    matA[i*matWidth + (*value)] = __float2half(1.0f);
}

/* Fill matrix with data value. */
__global__ void static gpu_fill_data(char *join_column, char *data_column, int matWidth_k, half *matA, size_t tupleNum, int attrType) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= tupleNum) return;

    int index = i * attrType;
    int *join_value = (int*)&join_column[index];
    int *data_value = (int*)&data_column[index];
    matA[i * matWidth_k + (*join_value)] = __float2half((float)(*data_value));
}

__global__ void static gpu_fill_gb(char *join_column, char *data_column, int matWidth_k, half *matA, size_t tupleNum, int attrType) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= tupleNum) return;

    int index = i * attrType;
    int *join_value = (int*)&join_column[index];
    int *data_value = (int*)&data_column[index];
    matA[(*data_value) * matWidth_k + (*join_value)] = __float2half(1.0f);
}

__global__ void static gpu_fill_data_transpose(char *join_column, char *data_column, int matWidth_n, half *matB, size_t tupleNum, int attrType) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= tupleNum) return;

    int index = i * attrType;
    int *join_value = (int*)&join_column[index];
    int *data_value = (int*)&data_column[index];
    matB[(*join_value) * matWidth_n + i] = __float2half((float)(*data_value));
}

/* Fill matrix with ones according to groupBy column in transpose format. */
__global__ void static gpu_fill_gb_transpose(char *join_column, char *data_column, int matWidth_n, half *matB, size_t tupleNum, int attrType) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= tupleNum) return;

    int index = i * attrType;
    int *join_value = (int*)&join_column[index];
    int *data_value = (int*)&data_column[index];
    matB[(*join_value) * matWidth_n + (*data_value)] = __float2half(1.0f);
}

/*
 * Fill ones matrix in transpose matrix format.
 */
__global__ void static gpu_fill_transpose(char *column, int matWidth, half *matB, size_t tupleNum, int attrType) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= tupleNum) return;

    int index = i * attrType;
    int *value   = (int*)&column[index];
    int pos = (*value)*tupleNum+i;
    matB[pos] = __float2half(1.0f);
}

/* Fill matrix in dense format for matrix multiplication */
__global__ void static microbenchmark(char *mat_i, char *mat_j, char *mat_val, int matWidth, half *mat, size_t tupleNum, int attrType) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i >= tupleNum) return;

    int index = i * attrType;
    int *row  = (int*)&mat_i[index]; 
    int *col  = (int*)&mat_j[index]; 
    int *val  = (int*)&mat_val[index];
    mat[(*row)*matWidth+(*col)] = __int2half_rn(*val);
}

__global__ void static microbenchmark_transpose(char *mat_i, char *mat_j, char *mat_val, int matWidth, half *mat, size_t tupleNum, int attrType) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i >= tupleNum) return;

    int index = i * attrType;
    int *row  = (int*)&mat_i[index]; 
    int *col  = (int*)&mat_j[index]; 
    int *val  = (int*)&mat_val[index];
    mat[(*col)*matWidth+(*row)] = __int2half_rn(*val);
}

__global__ void static outdegree_fill(char *column_val, half *mat, size_t tupleNum, int attrType) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i >= tupleNum) return;

    int index = i * attrType;
    //int *colIndex   = (int*)&column_idx[index];
    int *val        = (int*)&column_val[index];
    //printf("idx: %d\tval: %d\n", i*matWidth + (*colIndex), (*val));
    mat[(*val)] = __hadd(mat[(*val)], __int2half_rn(1));
}

#ifdef CUBLAS_HALF
__global__ void gpu_transpose(half *odata, const half *idata, int row, int col) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int x = index % col;
    int y = index / col;

    if (x < col && y < row) {
        odata[x*row + y] = idata[y*col + x];
    }
}
#elif CUBLAS
__global__ void gpu_transpose(float *odata, const float *idata, int row, int col) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int x = index % col;
    int y = index / col;

    if (x < col && y < row) {
        odata[x*row + y] = idata[y*col + x];
    }
}
#endif

__global__ void static pageRankAdd(float *mat, int n, float pageRankAlpha, int numNodes) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        if (mat[idx] > 1e-6)
        //if (__hgt(mat[idx], __float2half(1e-6))) // precision loss
            mat[idx] += (float)(1-pageRankAlpha)/numNodes;
            //mat[idx] += __float2half((1-pageRankAlpha)/numNodes);
    }
}

/* Convert input data from half to float type */
__global__ void static convertFp16ToFp32(float *out, half *in, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        out[idx] = __half2float(in[idx]);
    }
}

/* Convert input data from half to float type */
__global__ void static convertFp32ToFp16(half *out, float *in, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        out[idx] = __float2half(in[idx]);
    }
}

/* Convert input data from char to half type */
__global__ void static convertCharToFp16(half *out, char *in, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        out[idx] = __int2half_rn((int)in[idx]);
    }
}

__global__ void groupByCount(float *data, int n, int *gbCount) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        if (data[idx] > 0.000001) {
        //if (data[idx] > 0.001) {
            atomicAdd(gbCount, 1);
        }
    }
}

/* set the first column of the matrix to be 1.0 */
__host__ static void set_mask(float *mask, int height, int width) {
    for (int i = 0; i < height*width; i+=width) {
        mask[i] = 1.0;
    }
}

/* set the first row of the matrix to be 1.0 */
__host__ static void set_mask2(float *mask, int height, int width) {
    
    for (int i = 0; i < width; i++) {
        mask[i] = 1.0;
    }
    
}

__host__ static void setVector(float *vec, int n) {
    for (int i = 0; i < n; i++)
        vec[i] = 1.0;
}

__host__ static void setRed(short *red, int n) {
    for (int i = 0; i < n; i++)
        red[i] = (short)1;
}

/* Get column index from aggregate function for later data copy. */
__host__ static void getValIndex(struct joinNode *jNode, struct groupByNode *gb, int *lValIndex, int *rValIndex, int &lgbIndex, int &rgbIndex, int &dataColIndex) {

    for (int i = 0; i < jNode->leftOutputAttrNum; i++) {
        for (int j = 0; j < gb->numFuncExpCol; j++) {
            if (jNode->leftPos[i] == gb->funcExpColIndex[j]) {
                lValIndex[i] = jNode->leftOutputIndex[i];

                if (dataColIndex == -1)
                    dataColIndex = jNode->leftOutputIndex[i];
            }
            if (jNode->leftPos[i] == gb->groupByIndex[0]) {
                lgbIndex = 1;
            }
        }
    } 
    
    for (int i = 0; i < jNode->rightOutputAttrNum; i++) {
        for (int j = 0; j < gb->numFuncExpCol; j++) {
            if (jNode->rightPos[i] == gb->funcExpColIndex[j]) {
                rValIndex[i] = jNode->rightOutputIndex[i];

                if (dataColIndex == -1)
                    dataColIndex = jNode->rightOutputIndex[i];
            }
            if (jNode->rightPos[i] == gb->groupByIndex[0]) {
                rgbIndex = 1;
            }
        }
    } 
}

/* Match the first groupBy attribute, return 0 (left), 1 (right)*/
__host__ static int getGbLeftRight(struct joinNode *jNode, struct groupByNode *gb, int &gbConstant, int &gbLeftRight) {
    if (gbConstant == 1) return -1;
    
    for (int i = 0; i < jNode->leftOutputAttrNum; i++) {
        if (jNode->leftPos[i] == gb->groupByIndex[0]) {
            return 0;
        }
    } 
    
    for (int i = 0; i < jNode->rightOutputAttrNum; i++) {
        if (jNode->rightPos[i] == gb->groupByIndex[0]) {
            return 1;
        }
    } 
    return -1;
}

/* Mimic the max() in relational database. */
__host__ int getMaxVal(char *column, size_t tupleNum, int attrType) {
    int localMax = 0;

    for (int i = 0; i < tupleNum; i++) {
        int *val = (int*)&column[i*attrType];
        if (localMax < *val) {
            localMax = *val;
        }
    }
    return localMax;
}

/* Need to copy values to device */
__global__ void getMaxValGPU(char *column, size_t tupleNum, int attrType, int *maxVal) {
    __shared__ int sharedMax;

    if (threadIdx.x == 0) {
        sharedMax = 0;
    }
    __syncthreads();

    int localMax = 0;
    for (int i = threadIdx.x; i < tupleNum; i += blockDim.x) {
        int index = i * attrType;
        int *value   = (int*)&column[index];

        if (localMax < abs(*value)) {
            localMax = abs(*value);
        }
    }

    atomicMax(&sharedMax, localMax);
    __syncthreads();
    
    if (threadIdx.x == 0) {
        *maxVal = sharedMax;
    }
}

/*
 * tcuJoinn using NVIDIA's cuBLAS lib to perform matrix multiplication and aggregation.
 *
 * Prerequisites:
 *  1. the data to be joined can be fit into GPU device memory.
 *  2. dimension table is not compressed
 *  3. user know the matrix dimension (#uniq values)
 *  
 * Input:
 *  jNode: contains information about the two joined tables.
 *  pp: records statistics such as kernel execution time
 *  matrix_dim: matrix width (number of unique values)
 *  gb: contains groupby information
 *
 * Output:
 *  Number of join counts and groupBy count if query contains groupBy keyword.
 *
 * Assumptions:
 *
 * 1. Two joined table schemas are the same for the simplicity of query parser.
 * 2. For all demo cases, all column types are INT, only PageRank queries 
 *    contain constant variable such as alpha and number of nodes.
 * 3. To support complex customized queries, code_gen.py modification is required.
 */
struct tableNode * tcuJoin(struct joinNode *jNode, struct statistic *pp, 
        int *matrix_dim, struct groupByNode *gb)
{

    struct timespec tcu_start, tcu_end;
    struct timespec init_start, init_end;
    struct timespec fill_start, fill_end;
    struct timespec maskRED_start, maskRED_end;
    struct timespec pagerankVerify_start, pagerankVerify_end;
    struct timespec cuMemcpy_start, cuMemcpy_end;

    struct tableNode * res = NULL;
    int leftTupleNum = jNode->leftTable->tupleNum;
    int rightTupleNum = jNode->rightTable->tupleNum;
    uint64_t MATRIX_M, MATRIX_N, MATRIX_K; // avoid overflow

    MATRIX_K = *matrix_dim; // user input, matrix width(#unique values)
#ifdef MICRO
    //Note: In our matrix multiplication cases, square matrix(M=N=K) are used
    MATRIX_M = MATRIX_K;
    MATRIX_N = MATRIX_K;
#else
    MATRIX_M = (uint64_t)leftTupleNum;
    MATRIX_N = (uint64_t)rightTupleNum;
#endif // end of MICRO
    long foreignKeySize = jNode->leftTable->attrTotalSize[jNode->leftKeyIndex];
    long primaryKeySize = jNode->rightTable->attrTotalSize[jNode->rightKeyIndex];
    
    float pageRankAlpha;

    int gbConstant = 0;   // 0: has groupBy, 1: no groupBy keyword
    int gbLeftRight = -1; // 0: gb by left, 1: gb by right
    int gbMatWidth = 0;   // size of dom(gb_column.val)

    if (gb->groupByColNum == 1 && gb->groupByIndex[0] == -1) {
        gbConstant = 1;
    }
    
    // update MATRIX_M or MATRIX_N given groupBy keyword
    if (gbConstant != 1) { // contains groupBy keyword
        char *gb_column;
        // linear scan to find the max value of groupBy column 
        gbLeftRight = getGbLeftRight(jNode, gb, gbConstant, gbLeftRight);
        if (gbLeftRight == 0) {
            gb_column = jNode->leftTable->content[gb->groupByIndex[0]];

            gbMatWidth = getMaxVal(gb_column, jNode->leftTable->tupleNum, jNode->leftOutputAttrType[0]) + 1;
            printf("matA gbMatWidth: %d\n", gbMatWidth);
        } else if (gbLeftRight == 1) {
            gb_column = jNode->rightTable->content[gb->groupByIndex[0]];
            gbMatWidth = getMaxVal(gb_column, jNode->rightTable->tupleNum, jNode->rightOutputAttrType[0]) + 1;
            printf("matB gbMatWidth: %d\n", gbMatWidth);
            // update
            MATRIX_N = gbMatWidth;
        } else {
            printf("No matched column found.\n");
        }
    }

    // TODO: determine which table to copy value column (left/right or both)
    // column index are leftOutputIndex[0] or rightOutputIndex[0]
    printf("numFuncExpCol: %d\n", gb->numFuncExpCol); // determine number of data column to be copied

    int *lValIndex, *rValIndex;
    int dataColIndex = -1;
    int lgbIndex = -1, rgbIndex = -1;
    lValIndex = (int *)malloc(sizeof(int) * jNode->leftOutputAttrNum);
    rValIndex = (int *)malloc(sizeof(int) * jNode->rightOutputAttrNum);
    memset(lValIndex, -1, sizeof(int) * jNode->leftOutputAttrNum);
    memset(rValIndex, -1, sizeof(int) * jNode->rightOutputAttrNum);

    // get data value index from gbNode
    getValIndex(jNode, gb, lValIndex, rValIndex, lgbIndex, rgbIndex, dataColIndex);
    printf("lValIndex[0]: %d\n", lValIndex[0]); // data copy from lValIndex
    printf("rValIndex[0]: %d\n", rValIndex[0]);
    printf("lgbIndex: %d\n", lgbIndex); // data copy from lValIndex
    printf("rgbIndex: %d\n", rgbIndex);
    printf("dataColIndex: %d\n", dataColIndex);


#ifdef PAGERANK
    //printf("func: %d\n", gb->gbExp[0].func);
    //printf("PageRank constant: %.3f\n", ((struct mathExp *)((struct mathExp *) gb->gbExp[0].exp.exp)[0].exp)[0].consValue);
    pageRankAlpha = ((struct mathExp *)((struct mathExp *) gb->gbExp[0].exp.exp)[0].exp)[0].consValue;
    //printf("(1-alpha)/#node: %.6f\n", ((struct mathExp *) gb->gbExp[0].exp.exp)[1].consValue);
#endif
    

//#ifdef DEBUG
    //cudaPrintfInit();
//#endif
    clock_gettime(CLOCK_REALTIME, &tcu_start);
    clock_gettime(CLOCK_REALTIME, &init_start);

/*
    printf("Left Tuple #: %d\n", leftTupleNum);
    printf("Right Tuple #: %d\n", rightTupleNum);
    printf("MATRIX_M: %lu\n", MATRIX_M);
    printf("MATRIX_N: %lu\n", MATRIX_N);
    printf("MATRIX_K: %lu\n", MATRIX_K);
*/
#ifdef PAGERANK
    //printf("PageRank Alpha: %.3f\n", pageRankAlpha);
    //printf("(1-alpha)/#node: %.6f\n", (1-pageRankAlpha)/MATRIX_K);
#endif


#if defined(CUBLAS_HALF) || defined(CUBLAS)
    //struct timespec debug_start, debug_end; // hipblasCreate has init overhead
    struct timespec count_start, count_end;
    //struct timespec transpose_start, transpose_end;
#endif

    // read row data from tbl
    char *gpu_fact, *gpu_dim;         // joined column index
    char *gpu_fact_j, *gpu_dim_j;     // another index for dense table
    char *gpu_fact_val, *gpu_dim_val; // value
    char *gpu_ldata, *gpu_rdata;      // data columns of left/right tables
    char *d_redMat;
    half *d_redMatFp16;

    float alpha = 1.0f;
    float beta = 0.0f;
#ifdef CUBLAS_HALF
    half *d_fp16_A, *d_fp16_B, *d_fp16_BT;
//    half *d_fp16_A, *d_fp16_BT;
    float *c_cublas;
    half *c_fp16_cublas;

//    char *gpu_fact, *gpu_dim;         // raw data idx
//    char *gpu_fact_val, *gpu_dim_val; // raw data val
//    float alpha = 1.0f;
//    float beta = 0.0f;
    half alpha_fp16 = __float2half(1.0f);
    half beta_fp16 = __float2half(1.0f);
    float *c_host_cublas;

#ifdef PAGERANK
    //char *factID, *dimID; // use previous gpu_fact/gpu_dim
    char *factVal, *dimVal;
#endif

    struct timespec gbCount_start, gbCount_end;
    // TODO: move this into RED after decouple
    float *h_red, *d_red;
#ifdef RED
//    float *h_red, *d_red;
    float *h_red2, *d_red2;
#endif

    hipblasHandle_t cublasHandle;
    hipEvent_t startcublasEX;
    hipEvent_t stopcublasEX;

    cudaErrCheck(hipEventCreate(&startcublasEX));
    cudaErrCheck(hipEventCreate(&stopcublasEX));
    //clock_gettime(CLOCK_REALTIME, &debug_start);
    cublasErrCheck(hipblasCreate(&cublasHandle));
    //clock_gettime(CLOCK_REALTIME, &debug_end);
    cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH));
//    clock_gettime(CLOCK_REALTIME, &debug_start);
    //cublasErrCheck(hipblasCreate(&cublasHandle));
//    clock_gettime(CLOCK_REALTIME, &debug_end);
    //cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH));

#elif CUBLAS // SGEMM
    float *h_fp32_A, *h_fp32_B;             // host float32 array
    float *d_fp32_A, *d_fp32_B, *d_fp32_BT; // device float32 array
//    char *gpu_fact, *gpu_dim; // raw data index
//    char *gpu_fact_val, *gpu_dim_val; // raw data val
    float *c_sgemm, *c_host_sgemm;
//    float alpha = 1.0f;
//    float beta = 0.0f;

    hipblasHandle_t cublasHandle_default;
    hipEvent_t startcublas;
    hipEvent_t stopcublas;

    cudaErrCheck(hipEventCreate(&startcublas));
    cudaErrCheck(hipEventCreate(&stopcublas));
    //clock_gettime(CLOCK_REALTIME, &debug_start);
    cublasErrCheck(hipblasCreate(&cublasHandle_default));
    //clock_gettime(CLOCK_REALTIME, &debug_end);
    cublasErrCheck(hipblasSetMathMode(cublasHandle_default,HIPBLAS_DEFAULT_MATH));
#endif

// allocate device memory for inputs
#ifdef CUBLAS_HALF
    
//    long foreignKeySize = jNode->leftTable->attrTotalSize[jNode->leftKeyIndex];
//    long primaryKeySize = jNode->rightTable->attrTotalSize[jNode->rightKeyIndex];

    //printf("gpu_fact size: %d\tgpu_dim size: %d\n", foreignKeySize, primaryKeySize);
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpu_fact,foreignKeySize));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpu_dim,primaryKeySize));

    if (lValIndex[0] != -1 || lgbIndex != -1) {
        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpu_ldata,foreignKeySize));
        printf("hipMalloc left_data column\n");
    }

    if (rValIndex[0] != -1 || rgbIndex != -1) {
        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpu_rdata,primaryKeySize));
        printf("hipMalloc right_data column\n");
    }
#ifdef MICRO
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpu_fact_j,foreignKeySize));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpu_dim_j,primaryKeySize));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpu_fact_val,foreignKeySize));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpu_dim_val,primaryKeySize));
#endif

#ifdef PAGERANK // only for pagerank dataset
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&factVal,foreignKeySize));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&dimVal,primaryKeySize));
#endif

#ifdef OUTDEGREE
    // TODO:create square matrix for PageRank Q1 output
    // any row in the resulting matrix is the answer
    // B mat filling by counting src node -- B should be MATRIX_N * 1
    c_host_cublas = (float*)calloc(MATRIX_M*MATRIX_M, sizeof(float));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&c_cublas, (uint64_t)MATRIX_M * (uint64_t)MATRIX_M * sizeof(float)));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&d_fp16_A, (uint64_t)MATRIX_M * 1 * sizeof(half)));
    CUDA_SAFE_CALL_NO_SYNC(hipMemset(d_fp16_A, 1, (uint64_t)MATRIX_M * 1 * sizeof(half)));
    // same dimension as A mat
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&d_fp16_B, (uint64_t)MATRIX_M * 1 * sizeof(half)));
    CUDA_SAFE_CALL_NO_SYNC(hipMemset(d_fp16_B, 0, (uint64_t)MATRIX_M * 1 * sizeof(half)));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpu_fact_val,foreignKeySize));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpu_dim_val,primaryKeySize));

#else
    c_host_cublas = (float*)calloc(MATRIX_M*MATRIX_N, sizeof(float));
    //TODO: seems need to move hipMalloc into if-condition to dynamically adjust size
    printf("hipMalloc here\n");
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&c_cublas,(uint64_t)MATRIX_M*(uint64_t)MATRIX_N*sizeof(float)));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&d_fp16_A,(uint64_t)MATRIX_M*(uint64_t)MATRIX_K*sizeof(half)));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&d_fp16_B,(uint64_t)MATRIX_N*(uint64_t)MATRIX_K*sizeof(half)));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&d_fp16_BT,(uint64_t)MATRIX_K*(uint64_t)MATRIX_N*sizeof(half)));
#endif //end of OUTDEGREE

    // TODO: move this into RED after decouple
    h_red = (float*)calloc(MATRIX_N, sizeof(float));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&d_red, MATRIX_N * sizeof(float)));
#ifdef RED
//    h_red = (float*)calloc(MATRIX_N, sizeof(float));
//    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&d_red, MATRIX_N * sizeof(float)));
    h_red2 = (float*)calloc(MATRIX_M, sizeof(float));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&d_red2, MATRIX_M * sizeof(float)));
#endif

#elif CUBLAS

#ifdef MICRO
    long foreignKeySize = jNode->leftTable->attrTotalSize[jNode->leftKeyIndex];
    long primaryKeySize = sizeof(int) * jNode->rightTable->tupleNum;

    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpu_fact,foreignKeySize));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpu_dim,primaryKeySize));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpu_fact_j,foreignKeySize));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpu_dim_j,primaryKeySize));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpu_fact_val,foreignKeySize));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpu_dim_val,primaryKeySize));
#endif // MICRO

    h_fp32_A =     (float*)calloc(MATRIX_M*MATRIX_K, sizeof(float));
    h_fp32_B =     (float*)calloc(MATRIX_N*MATRIX_K, sizeof(float));
    c_host_sgemm = (float*)calloc(MATRIX_M*MATRIX_N, sizeof(float));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&d_fp32_A, MATRIX_M * MATRIX_K * sizeof(float)));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&d_fp32_B, MATRIX_N * MATRIX_K * sizeof(float)));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&d_fp32_BT, MATRIX_K * MATRIX_N * sizeof(float)));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&c_sgemm, MATRIX_M * MATRIX_N * sizeof(float)));
#endif // end of initialization
    clock_gettime(CLOCK_REALTIME, &init_end);

#ifdef CUBLAS_HALF
// call different matrix filling methods according to dataset
#ifdef MICRO
    clock_gettime(CLOCK_REALTIME, &cuMemcpy_start);

    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpu_fact,jNode->leftTable->content[jNode->leftKeyIndex], foreignKeySize,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpu_dim,jNode->rightTable->content[jNode->rightKeyIndex], primaryKeySize,hipMemcpyHostToDevice));
    // ystree.py gen_column_index generates index for select_list first
    // joined attr with -1 index which means the last index
    // other attr indices follow the sequence without certain getter function to access
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpu_fact_j,jNode->leftTable->content[0], foreignKeySize,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpu_dim_j,jNode->rightTable->content[0], primaryKeySize,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpu_fact_val,jNode->leftTable->content[1], foreignKeySize,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpu_dim_val,jNode->rightTable->content[1], primaryKeySize,hipMemcpyHostToDevice));
    clock_gettime(CLOCK_REALTIME, &cuMemcpy_end);

    clock_gettime(CLOCK_REALTIME, &fill_start); 
    microbenchmark<<<(MAX_THREADS+leftTupleNum-1)/MAX_THREADS,MAX_THREADS>>> (gpu_fact,
            gpu_fact_j,
            gpu_fact_val,
            MATRIX_K,
            d_fp16_A,
            leftTupleNum,
            jNode->leftTable->attrType[jNode->leftKeyIndex]);
    cudaErrCheck(hipFree(gpu_fact));
    cudaErrCheck(hipFree(gpu_fact_j));
    cudaErrCheck(hipFree(gpu_fact_val));
    microbenchmark_transpose<<<(MAX_THREADS+rightTupleNum-1)/MAX_THREADS,MAX_THREADS>>> (gpu_dim,
            gpu_dim_j,
            gpu_dim_val,
            MATRIX_K,
            d_fp16_BT,
            rightTupleNum,
            jNode->rightTable->attrType[jNode->rightKeyIndex]);
    cudaErrCheck(hipFree(gpu_dim));
    cudaErrCheck(hipFree(gpu_dim_j));
    cudaErrCheck(hipFree(gpu_dim_val));

    clock_gettime(CLOCK_REALTIME, &fill_end); 
#elif OUTDEGREE // PageRank Q1

    clock_gettime(CLOCK_REALTIME, &cuMemcpy_start);
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpu_fact_val,jNode->leftTable->content[1], foreignKeySize,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpu_dim_val,jNode->rightTable->content[1], primaryKeySize,hipMemcpyHostToDevice));
    clock_gettime(CLOCK_REALTIME, &cuMemcpy_end);

    clock_gettime(CLOCK_REALTIME, &fill_start); 
    outdegree_fill<<<(MAX_THREADS+leftTupleNum-1)/MAX_THREADS,MAX_THREADS>>> (gpu_fact_val,
            d_fp16_A,
            leftTupleNum,
            jNode->leftTable->attrType[jNode->leftKeyIndex]);
    outdegree_fill<<<(MAX_THREADS+rightTupleNum-1)/MAX_THREADS,MAX_THREADS>>> (gpu_dim_val,
            d_fp16_B,
            rightTupleNum,
            jNode->rightTable->attrType[jNode->rightKeyIndex]);
    clock_gettime(CLOCK_REALTIME, &fill_end); 
// end of OUTDEGREE to hipMemcpy and filling matrix

#else //  MM for join count

    // hipMemcpyHostToDevice raw data->char *column
    clock_gettime(CLOCK_REALTIME, &cuMemcpy_start);
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpu_fact,jNode->leftTable->content[jNode->leftKeyIndex], foreignKeySize,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpu_dim,jNode->rightTable->content[jNode->rightKeyIndex], primaryKeySize,hipMemcpyHostToDevice));
    if (lValIndex[0] != -1 || lgbIndex != -1) {
        CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpu_ldata,jNode->leftTable->content[dataColIndex], foreignKeySize,hipMemcpyHostToDevice));
        printf("hipMemcpy gpu_ldata\n");
    }
    if (rValIndex[0] != -1 || rgbIndex != -1) {
        CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpu_rdata,jNode->rightTable->content[dataColIndex], primaryKeySize,hipMemcpyHostToDevice));
        printf("hipMemcpy gpu_rdata\n");
    }

    /*for (int i = 0; i < 5; i ++) {
        int *value   = (int*)&jNode->leftTable->content[0][i*4];
        printf("%d\n", *value);
    }*/


#ifdef PAGERANK  // pagerank requires additional float value instead of filling 0/1
    int factCol = jNode->leftOutputIndex[jNode->leftOutputAttrNum-1];
    int dimCol = jNode->rightOutputIndex[jNode->rightOutputAttrNum-1];
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(factVal,jNode->leftTable->content[factCol], foreignKeySize,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(dimVal,jNode->rightTable->content[dimCol], primaryKeySize,hipMemcpyHostToDevice));
#endif // end of PageRank
    clock_gettime(CLOCK_REALTIME, &cuMemcpy_end);

    clock_gettime(CLOCK_REALTIME, &fill_start);  // filling time (except for MICRO, OUTDEGREE)
#ifdef PAGERANK // specifically design for PageRank

    if (gb->gbExp[0].func == SUM) { // 20, defined in common.h
        pagerank<<<(MAX_THREADS+leftTupleNum-1)/MAX_THREADS,MAX_THREADS>>> (gpu_fact,
                factVal,
                MATRIX_K,
                d_fp16_A,
                leftTupleNum,
                jNode->leftTable->attrType[jNode->leftKeyIndex],
                jNode->leftTable->attrType[factCol],
                pageRankAlpha); 
        cudaErrCheck(hipFree(gpu_fact));
        cudaErrCheck(hipFree(factVal));

        pagerank<<<(MAX_THREADS+rightTupleNum-1)/MAX_THREADS,MAX_THREADS>>> (gpu_dim,
                dimVal,
                MATRIX_K,
//                d_fp16_B,
                d_fp16_BT,
                rightTupleNum,
                jNode->rightTable->attrType[jNode->rightKeyIndex],
                jNode->rightTable->attrType[dimCol],
                1.0); 
        cudaErrCheck(hipFree(gpu_dim));
        cudaErrCheck(hipFree(dimVal));

        CUDA_SAFE_CALL_NO_SYNC(hipMemset(c_cublas,0,(uint64_t)MATRIX_M*(uint64_t)MATRIX_N*sizeof(float)));
    }
#else // query ask for join counts except for OUTDEGREE, MICRO, PAGERANK
    //TODO: call corresponding filling method (check SQL pattern)
    if (gb->gbExp[gb->aggFuncIndex].func == SUM) {
        //printf("Query contains SUM\n");

        if (gb->numFuncExpCol == 1) { // Q3
            // judge whether to pass left or right data column
           // printf("rValIndex[0]\n", rValIndex[0]);
           // printf("lValIndex[0]\n", lValIndex[0]);
            if (rValIndex[0] == -1) // pass left 
            {
               // getMaxValGPU(char *column, size_t tupleNum, int attrType, int *maxVal);
                // gpu_fill_data (left), gpu_fill_transpose (right)
                gpu_fill_data<<<(MAX_THREADS+leftTupleNum-1)/MAX_THREADS,MAX_THREADS>>> (gpu_fact,
                    gpu_ldata,    
                    MATRIX_K,
                    d_fp16_A,
                    leftTupleNum,
                    jNode->leftTable->attrType[jNode->leftKeyIndex]);

                cudaErrCheck(hipFree(gpu_fact));
                cudaErrCheck(hipFree(gpu_ldata));
                // TODO: right fill with ones_gb
                
                gpu_fill_gb_transpose<<<(MAX_THREADS+rightTupleNum-1)/MAX_THREADS,MAX_THREADS>>> (
                        gpu_dim,
                        gpu_rdata,
                        MATRIX_N,
                        d_fp16_BT,
                        rightTupleNum,
                        jNode->rightTable->attrType[jNode->rightKeyIndex]);

                cudaErrCheck(hipFree(gpu_dim));
                cudaErrCheck(hipFree(gpu_rdata));

                /*
                gpu_fill_transpose<<<(MAX_THREADS+rightTupleNum-1)/MAX_THREADS,MAX_THREADS>>> (gpu_dim,
                    MATRIX_K,
                    d_fp16_BT,
                    rightTupleNum,
                    jNode->rightTable->attrType[jNode->rightKeyIndex]);

                cudaErrCheck(hipFree(gpu_dim));
                */
            } 
            else if (lValIndex[0] == -1)// pass right 
            {
                // matA -> gbMatWidth x MATRIX_K
                gpu_fill_gb<<<(MAX_THREADS+leftTupleNum-1)/MAX_THREADS,MAX_THREADS>>> (
                        gpu_fact,
                        gpu_ldata,    
                        MATRIX_K,
                        d_fp16_A,
                        leftTupleNum,
                        jNode->leftTable->attrType[jNode->leftKeyIndex]);
                cudaErrCheck(hipFree(gpu_fact));
                cudaErrCheck(hipFree(gpu_ldata));

                // MATRIX_K x MATRIX_N
                gpu_fill_data_transpose<<<(MAX_THREADS+rightTupleNum-1)/MAX_THREADS,MAX_THREADS>>> (
                        gpu_dim,
                        gpu_rdata,
                        MATRIX_N,
                        d_fp16_BT,
                        rightTupleNum,
                        jNode->rightTable->attrType[jNode->rightKeyIndex]);
                cudaErrCheck(hipFree(gpu_dim));
                cudaErrCheck(hipFree(gpu_rdata));

            }
        }
        else if (gb->numFuncExpCol == 2) { // Q4, gb->numFuncExpCol == 2

        }
        


    }

    /* If has groupBy, after MM, then compute gbCount */
/*    if (gbConstant != 1) { // print gbCount
        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&d_redMat, 1 * MATRIX_M * sizeof(char)));
        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&d_redMatFp16, 1 * MATRIX_M * sizeof(half)));
        CUDA_SAFE_CALL_NO_SYNC(hipMemset(d_redMat, 1, MATRIX_M * sizeof(char)));
        convertCharToFp16 <<< (MATRIX_M + 255) / 256, 256 >>> (d_redMatFp16, 
                d_redMat, MATRIX_M);
        
        // compute groupBy count by performing reduction

    }*/

    /*
     Q3
     Need to determine take left/right data column and groupBy which table? => gbLeftRight
     0: left, 1: right
     1 -- either lValIndex or rValIndex is -1, one as actual value, the other as 1
     */
    /*
    if (gb->numFuncExpCol == 1) {
        // judge whether to pass left or right data column
        if (rValIndex[0] == -1) // pass left 
        {
            // gpu_fill_data (left), gpu_fill_transpose (right)

        } 
        else if (lValIndex[0] == -1)// pass right 
        {
            // call gpu_fill (left), gpu_fill_transpose_data (right)

        }
    }*/

    /*
     Q4
     if (gb->math_op == MULTIPLY && (lValIndex[0] != -1 && rValIndex[0] != -1))
     both lValIndex/rValIndex are not -1, all pass value into func 
     */

    /*
     Else case -- leave it with the general matrix multiplication => return join_count
     */

    /*
    gpu_fill<<<(MAX_THREADS+leftTupleNum-1)/MAX_THREADS,MAX_THREADS>>> (gpu_fact,
            MATRIX_K,
            d_fp16_A,
            leftTupleNum,
            jNode->leftTable->attrType[jNode->leftKeyIndex]);
    cudaErrCheck(hipFree(gpu_fact));
    
    gpu_fill_transpose<<<(MAX_THREADS+rightTupleNum-1)/MAX_THREADS,MAX_THREADS>>> (gpu_dim,
            MATRIX_K,
            d_fp16_BT,
            rightTupleNum,
            jNode->rightTable->attrType[jNode->rightKeyIndex]);
    cudaErrCheck(hipFree(gpu_dim));
    */
    
#endif
    clock_gettime(CLOCK_REALTIME, &fill_end); 

#endif // end of fill matrix for CUBLAS_HALF

#elif CUBLAS

#ifdef MICRO
    //int A_tupleNum = jNode->leftTable->tupleNum;
    //int B_tupleNum = jNode->rightTable->tupleNum;

    clock_gettime(CLOCK_REALTIME, &cuMemcpy_start);
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpu_fact,jNode->leftTable->content[jNode->leftKeyIndex], foreignKeySize,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpu_dim,jNode->rightTable->content[jNode->rightKeyIndex], primaryKeySize,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpu_fact_j,jNode->leftTable->content[0], foreignKeySize,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpu_dim_j,jNode->rightTable->content[0], primaryKeySize,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpu_fact_val,jNode->leftTable->content[1], foreignKeySize,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpu_dim_val,jNode->rightTable->content[1], primaryKeySize,hipMemcpyHostToDevice));
    clock_gettime(CLOCK_REALTIME, &cuMemcpy_end);

    clock_gettime(CLOCK_REALTIME, &fill_start); 
    microbenchmark<<<(MAX_THREADS+leftTupleNum-1)/MAX_THREADS,MAX_THREADS>>> (gpu_fact,
            gpu_fact_j,
            gpu_fact_val,
            MATRIX_K,
            d_fp32_A,
            leftTupleNum,
            jNode->leftTable->attrType[jNode->leftKeyIndex]);
    cudaErrCheck(hipFree(gpu_fact));
    microbenchmark_transpose<<<(MAX_THREADS+rightTupleNum-1)/MAX_THREADS,MAX_THREADS>>> (gpu_dim,
            gpu_dim_j,
            gpu_dim_val,
            MATRIX_K,
            d_fp32_BT,
            rightTupleNum,
            jNode->rightTable->attrType[jNode->rightKeyIndex]);
    cudaErrCheck(hipFree(gpu_dim));

    clock_gettime(CLOCK_REALTIME, &fill_end); 
#else
    // No other modes for now

#endif // end of MICRO

#endif // all modes (CUBLAS_HALF and CUBLAS) filling matrix end

// set up mask for reduction if required
clock_gettime(CLOCK_REALTIME, &maskRED_start); 
#ifdef CUBLAS_HALF

#ifdef MICRO
// do nothing

#else

    // TODO: move this into RED after decouple
    setVector(h_red, MATRIX_N);
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(d_red, h_red, sizeof(float) * MATRIX_N, hipMemcpyHostToDevice));
#ifdef RED
//    setVector(h_red, MATRIX_N);
//    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(d_red, h_red, sizeof(float) * MATRIX_N, hipMemcpyHostToDevice));
    setVector(h_red2, MATRIX_M);
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(d_red2, h_red2, sizeof(float) * MATRIX_M, hipMemcpyHostToDevice));
#endif

#endif // end of else MICRO
#endif // end of setting mask for CUBLAS_HALF
clock_gettime(CLOCK_REALTIME, &maskRED_end); 

// transpose B matrix
#ifdef CUBLAS_HALF

#ifdef OUTDEGREE
    //outdegree doesn't need to transpose
#elif MICRO
    //microbenchmark doesn't need to transpose
#else    
    //clock_gettime(CLOCK_REALTIME, &transpose_start);
    // no need to transpose if use gpu_fill_transpose
//    gpu_transpose<<< (MATRIX_N * MATRIX_K + 255) / 256, 256 >>> (d_fp16_BT, d_fp16_B, MATRIX_N, MATRIX_K);
    //clock_gettime(CLOCK_REALTIME, &transpose_end);
//    cudaErrCheck(hipFree(d_fp16_B));
#endif // end of OUTDEGREE -- transpose B matrix

#elif CUBLAS

    //clock_gettime(CLOCK_REALTIME, &transpose_start);
    //TODO: call gpu_fill_transpose directly
    //gpu_transpose<<< (MATRIX_N * MATRIX_K + 255) / 256, 256 >>> (d_fp32_BT, d_fp32_B, MATRIX_N, MATRIX_K);
    //clock_gettime(CLOCK_REALTIME, &transpose_end);
    //cudaErrCheck(hipFree(d_fp32_B));
#endif // end of transposing B matrix

// start cublasGemm lib to perform MM
#ifdef CUBLAS_HALF

#ifdef MICRO
    printf("Running Matrix Multiplication (dense matrix) using GemmEx with TCUs...\n");
    cudaErrCheck(hipEventRecord(startcublasEX));
    cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                MATRIX_N, MATRIX_M, MATRIX_K,
                &alpha,
                d_fp16_A, HIP_R_16F, MATRIX_N,
                d_fp16_BT, HIP_R_16F, MATRIX_K,
                &beta,
                c_cublas, HIP_R_32F, MATRIX_N,
                HIPBLAS_COMPUTE_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP)); // CUDA 11
    cudaErrCheck(hipFree(d_fp16_A));
    cudaErrCheck(hipFree(d_fp16_B));

#elif OUTDEGREE
    printf("Running compute_outdegree with GemmEx with TCUs...\n");
    cudaErrCheck(hipEventRecord(startcublasEX));
    cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                MATRIX_M, MATRIX_M, 1,
                &alpha,
                d_fp16_B, HIP_R_16F, MATRIX_M,
                d_fp16_A, HIP_R_16F, 1,
                &beta,
                c_cublas, HIP_R_32F, MATRIX_M,
                HIPBLAS_COMPUTE_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP)); // CUDA 11
    cudaErrCheck(hipFree(d_fp16_A));
    cudaErrCheck(hipFree(d_fp16_B));

// end of OUTDEGREE tcu cublasEX
#else

    float *red_sum;
    int *gbCount;
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&red_sum, MATRIX_M * sizeof(float)));
    if (gb && gb->gbExp[1].func == COUNT) {
        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gbCount, sizeof(int)));
        CUDA_SAFE_CALL_NO_SYNC(hipMemset(gbCount, 0, sizeof(int)));
    }

#ifdef RED
    float *red_sum2;
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&red_sum2, 1 * sizeof(float)));
#endif
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&c_fp16_cublas, (uint64_t)MATRIX_M * (uint64_t)MATRIX_N * sizeof(half)));
    //printf("Running with cuBLAS on TCUs...\n");
    cudaErrCheck(hipEventRecord(startcublasEX));
/*    cublasErrCheck(hipblasHgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                MATRIX_N, MATRIX_M, MATRIX_K,
                &alpha_fp16,
                d_fp16_BT,MATRIX_N,
                d_fp16_A,MATRIX_K,
                &beta_fp16,
                c_fp16_cublas, MATRIX_N));*/
#ifdef RED    
    printf("Running GemmEx RED on TCUs...\n");
    cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                MATRIX_N, MATRIX_M, MATRIX_K,
                &alpha,
                d_fp16_BT, HIP_R_16F, MATRIX_N,
                d_fp16_A, HIP_R_16F, MATRIX_K,
                &beta,
                c_cublas, HIP_R_32F, MATRIX_N,
                //HIP_R_32F, CUBLAS_GEMM_DFALT_TENSOR_OP)); // tcu
                HIPBLAS_COMPUTE_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP)); // CUDA 11
    cudaErrCheck(hipFree(d_fp16_A));
    cudaErrCheck(hipFree(d_fp16_BT));
#else
    // NOTE: YDB's groupby is not group by clause but aggregate function
    // outdegree.sql, gb->gbExp[0].func == DESC
    if (gb && gb->gbExp[1].func == COUNT) {
        printf("Running GemmEx COUNT on TCUs...\n");
        cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                MATRIX_N, MATRIX_M, MATRIX_K,
                &alpha,
                d_fp16_BT, HIP_R_16F, MATRIX_N,
                d_fp16_A, HIP_R_16F, MATRIX_K,
                &beta,
                c_cublas, HIP_R_32F, MATRIX_N,
                //HIP_R_32F, CUBLAS_GEMM_DFALT_TENSOR_OP)); // tcu
                HIPBLAS_COMPUTE_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP)); // CUDA 11
        cudaErrCheck(hipFree(d_fp16_A));
        cudaErrCheck(hipFree(d_fp16_BT));

        clock_gettime(CLOCK_REALTIME, &gbCount_start);
        cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                1, MATRIX_M, MATRIX_N,
                &alpha,
                d_red, HIP_R_32F, 1,
                c_cublas, HIP_R_32F, MATRIX_N,
                &beta,
                red_sum, HIP_R_32F, 1,
                //HIP_R_32F, CUBLAS_GEMM_DFALT_TENSOR_OP)); // tcu
                HIPBLAS_COMPUTE_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP)); // CUDA 11

        
        // implements COUNT operation -- print node.id with outdegree cnt
        /*
        if (gb->gbExp[1].func == COUNT)
            count_op<<<(MATRIX_M + 255) / 256, 256>>> (red_sum, MATRIX_M);
        */
        
        //clock_gettime(CLOCK_REALTIME, &gbCount_start);
        gb_count<<<(MATRIX_M + 255) / 256, 256>>> (red_sum, MATRIX_M, gbCount);
        clock_gettime(CLOCK_REALTIME, &gbCount_end);
    } else if (gb && gb->gbExp[0].func == SUM) { // no group by clause, only SUM
        
        /*
        cublasErrCheck(hipblasHgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                MATRIX_N, MATRIX_M, MATRIX_K,
                &alpha_fp16,
                d_fp16_BT,MATRIX_N,
                d_fp16_A,MATRIX_K,
                &beta_fp16,
                c_fp16_cublas, MATRIX_N));
        */
        //printf("MxK:%dx%d\tKxN:%dx%d\n", MATRIX_M, MATRIX_K, MATRIX_K, gbMatWidth);
        printf("MxK:%dx%d\tKxN:%dx%d\n", MATRIX_M, MATRIX_K, MATRIX_K, MATRIX_N);
        // TODO: have a logic to judge left/right?

        printf("Running GemmEx (Group-by aggregates) on TCUs...\n");
        cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                MATRIX_N, MATRIX_M, MATRIX_K,
                &alpha,
                d_fp16_BT, HIP_R_16F, MATRIX_N,
                d_fp16_A, HIP_R_16F, MATRIX_K,
                &beta,
                c_cublas, HIP_R_32F, MATRIX_N,
                HIPBLAS_COMPUTE_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP));
        cudaErrCheck(hipFree(d_fp16_A));
        cudaErrCheck(hipFree(d_fp16_BT));
        
        // If has groupBy, return gbCount after MM /
        if (gbConstant != 1) {
            CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&d_redMat, 1 * MATRIX_M * sizeof(char)));
            CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&d_redMatFp16, 1 * MATRIX_M * sizeof(half)));
            CUDA_SAFE_CALL_NO_SYNC(hipMemset(d_redMat, 1, MATRIX_M * sizeof(char)));
            convertCharToFp16 <<< (MATRIX_M + 255) / 256, 256 >>> (d_redMatFp16, 
                d_redMat, MATRIX_M);
        
            //TODO: compute groupBy count by performing reduction
            half *temp_c;
            //CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&temp_c, MATRIX_M * gbMatWidth * sizeof(half)));
            CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&temp_c, MATRIX_M * MATRIX_N * sizeof(half)));
            
            //convertFp32ToFp16 <<< (MATRIX_M * gbMatWidth + 255) / 256, 256 >>> (temp_c, c_cublas, MATRIX_M * gbMatWidth);
            convertFp32ToFp16 <<< (MATRIX_M * MATRIX_N + 255) / 256, 256 >>> (temp_c, c_cublas, MATRIX_M * MATRIX_N);

            float *d_reduction_res;
            CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&d_reduction_res, 1 * MATRIX_N * sizeof(float)));

            printf("Perform groupBy reduction...\n");
            cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                    //gbMatWidth, 1, MATRIX_M,
                    MATRIX_N, 1, MATRIX_M,
                    &alpha,
                    //temp_c, HIP_R_16F, gbMatWidth,
                    temp_c, HIP_R_16F, MATRIX_N,
                    d_redMatFp16, HIP_R_16F, MATRIX_M,
                    &beta,
                    //d_reduction_res, HIP_R_32F, gbMatWidth,
                    d_reduction_res, HIP_R_32F, MATRIX_N,
                    HIPBLAS_COMPUTE_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP));

            cudaErrCheck(hipFree(temp_c));
            cudaErrCheck(hipFree(d_redMatFp16));

            // count number of column with values
            int *d_gbCount, *h_gbCount;
            CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&d_gbCount, 1 * sizeof(int)));
            CUDA_SAFE_CALL_NO_SYNC(hipMemset(d_gbCount, 0, 1 * sizeof(int)));
            h_gbCount = (int*)malloc(1 * sizeof(int));
            
            //groupByCount<<<(gbMatWidth+255), 256>>> (d_reduction_res, gbMatWidth, d_gbCount);
            groupByCount<<<(MATRIX_N+255), 256>>> (d_reduction_res, MATRIX_N, d_gbCount);
            cudaErrCheck(hipFree(d_reduction_res));
            cudaErrCheck(hipMemcpy(h_gbCount, d_gbCount, 1 * sizeof(int), hipMemcpyDeviceToHost));
            printf("GroupBy Count: %d\n", *h_gbCount);

        }

//        pageRankAdd<<< (MATRIX_M * MATRIX_N + 255) / 256, 256 >>> (c_cublas, MATRIX_M*MATRIX_N, pageRankAlpha, MATRIX_K);
        // verify result
        /*
        cudaErrCheck(hipMemcpy(c_host_cublas, c_cublas, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToHost));
        verify_result(c_host_cublas, MATRIX_M, MATRIX_N);
        */  
    }
    else if (gbConstant !=1) { // contains groupBy keyword

        float *test_red;
        int *gbcount;
        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&test_red, MATRIX_M * sizeof(float)));
        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&gbcount, sizeof(int)));
        CUDA_SAFE_CALL_NO_SYNC(hipMemset(gbcount, 0, sizeof(int)));

        printf("Running GemmEX (w/ groupBy...)\n");
        // call TCU join operator
        cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                MATRIX_N, MATRIX_M, MATRIX_K,
                &alpha,
                d_fp16_BT, HIP_R_16F, MATRIX_N,
                d_fp16_A, HIP_R_16F, MATRIX_K,
                &beta,
                c_cublas, HIP_R_32F, MATRIX_N,
                //HIP_R_32F, CUBLAS_GEMM_DFALT_TENSOR_OP)); // tcu
                HIPBLAS_COMPUTE_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP)); // CUDA 11
        cudaErrCheck(hipFree(d_fp16_A));
        cudaErrCheck(hipFree(d_fp16_BT));

        cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                1, MATRIX_M, MATRIX_N,
                &alpha,
                d_red, HIP_R_32F, 1,
                c_cublas, HIP_R_32F, MATRIX_N,
                &beta,
                test_red, HIP_R_32F, 1,
                //HIP_R_32F, CUBLAS_GEMM_DFALT_TENSOR_OP)); // tcu
                HIPBLAS_COMPUTE_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP)); // CUDA 11

        // TODO: need to reduce one time (groupBy) then count 
//        hipblasStatus_t ret;
//        ret = hipblasCreate(&cublasHandle);
//        float *cb_res = (float*)malloc(sizeof(float));
//        ret = hipblasSasum(cublasHandle, MATRIX_M*MATRIX_N, c_cublas, 1, cb_res);
//        printf("groupBy count: %.0f\n", *cb_res);
        // call TCU groupBy operator => return gbCount
        printf("MATRIX_M: %d\n", MATRIX_M);
        gb_count<<<(MAX_THREADS+MATRIX_M-1)/MAX_THREADS,MAX_THREADS>>> (test_red, MATRIX_M, gbcount);

        int h_gbCount = 0;
        cudaErrCheck(hipMemcpy(&h_gbCount, gbcount, sizeof(int), hipMemcpyDeviceToHost));
        printf("groupBy count: %d\n", h_gbCount);

    } 
    else {
        printf("Running Hgemm...\n");
        // no group by keyword, directly perform hipblasHgemm
        cublasErrCheck(hipblasHgemm(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                MATRIX_N, MATRIX_M, MATRIX_K,
                &alpha_fp16,
                d_fp16_BT,MATRIX_N,
                d_fp16_A,MATRIX_K,
                &beta_fp16,
                c_fp16_cublas, MATRIX_N));
        cudaErrCheck(hipFree(d_fp16_A));
        cudaErrCheck(hipFree(d_fp16_BT));
    }
//    res->attrTotalSize[2] = 4*MATRIX_M*MATRIX_N;
#endif // end of RED TCU operation

#endif // end of MICRO -- L727
    cudaErrCheck(hipEventRecord(stopcublasEX));

#ifdef RED // reduction to get correct join counts
    clock_gettime(CLOCK_REALTIME, &gbCount_start);
    // 1st reduction -> single column
    cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                1, MATRIX_M, MATRIX_N,
                &alpha,
                d_red, HIP_R_32F, 1,
                c_cublas, HIP_R_32F, MATRIX_N,
                &beta,
                red_sum, HIP_R_32F, 1,
                //HIP_R_32F, CUBLAS_GEMM_DFALT_TENSOR_OP)); // tcu
                HIPBLAS_COMPUTE_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP)); // CUDA 11

    // return groupBy count
    if (gb && gb->gbExp[1].func == COUNT) {
        // FIXME: why red_sum is 0, gb_count check 0.0 or 0?
        gb_count<<<(MATRIX_M + 255) / 256, 256>>> (red_sum, MATRIX_M, gbCount);
    }
    
    // 2nd reduction -> sinlge value
    // return join count
    cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                1, 1, MATRIX_M,
                &alpha,
                red_sum, HIP_R_32F, 1,
                d_red2, HIP_R_32F, MATRIX_M,
                &beta,
                red_sum2, HIP_R_32F, 1,
                //HIP_R_32F, CUBLAS_GEMM_DFALT_TENSOR_OP)); // tcu
                HIPBLAS_COMPUTE_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP)); // CUDA 11
    clock_gettime(CLOCK_REALTIME, &gbCount_end);
    cudaErrCheck(hipFree(red_sum));
    cudaErrCheck(hipFree(d_red2));
    
#endif
//    cudaErrCheck(hipEventRecord(stopcublasEX));
#elif CUBLAS
    printf("Running with SGemm...\n");
    cudaErrCheck(hipEventRecord(startcublas));
    hipblasSgemm(cublasHandle_default, HIPBLAS_OP_N, HIPBLAS_OP_N,
            MATRIX_N, MATRIX_M, MATRIX_K,
            &alpha,
            d_fp32_BT, MATRIX_N,
            d_fp32_A, MATRIX_K,
            &beta,
            c_sgemm, MATRIX_N);
    cudaErrCheck(hipEventRecord(stopcublas));
    cudaErrCheck(hipFree(d_fp32_A));
    cudaErrCheck(hipFree(d_fp32_BT));
#endif    

#ifdef CUBLAS_HALF
    float cublasEXTime;

    cudaErrCheck(hipEventSynchronize(stopcublasEX));
    cudaErrCheck(hipEventElapsedTime(&cublasEXTime, startcublasEX, stopcublasEX));

    // test output
    /*
    float *tmp_res;
    tmp_res = (float*)calloc(MATRIX_M*MATRIX_N, sizeof(float));
    cudaErrCheck(hipMemcpy(tmp_res, c_cublas, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToHost));
    for (int i = 0; i < MATRIX_M*MATRIX_N; i++) {
        printf("%.2f\t", tmp_res[i]);
        if ((i+1)%MATRIX_N == 0)
            printf("\n\n");
    }
    */

#ifdef RED
    clock_gettime(CLOCK_REALTIME, &count_start);
    float *ans;
    ans = (float*)calloc(1, sizeof(float));
    cudaErrCheck(hipMemcpy(ans, red_sum2, 1 * sizeof(float), hipMemcpyDeviceToHost));
    if (gb && (gb->gbExp[1].func == COUNT || gb->gbExp[0].func == COUNT)) {
        int h_gbCount = 0;
        cudaErrCheck(hipMemcpy(&h_gbCount, gbCount, sizeof(int), hipMemcpyDeviceToHost));
        printf("groupBy count: %d\n", h_gbCount);
        double gbCount_elapse = (gbCount_end.tv_sec -  gbCount_start.tv_sec)* BILLION + gbCount_end.tv_nsec - gbCount_start.tv_nsec;
        printf("GroupBy Time: %lf(ms)\n", gbCount_elapse/(1000*1000));
    }
    clock_gettime(CLOCK_REALTIME, &count_end);
    printf("c_host_cublas reduction sum: %.0f\n", ans[0]);
    free(ans);
//    cudaErrCheck(hipFree(red_sum));
    cudaErrCheck(hipFree(red_sum2));
#elif PAGERANK
    // print is time consuming, hipMemcpy time is also for the purpose of verification
    clock_gettime(CLOCK_REALTIME, &pagerankVerify_start);
    cudaErrCheck(hipMemcpy(c_host_cublas, c_cublas, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToHost));
    clock_gettime(CLOCK_REALTIME, &pagerankVerify_end);
#elif MICRO
    // only for verification
    //cudaErrCheck(hipMemcpy(c_host_cublas, c_cublas, sizeof(float)*MATRIX_M*MATRIX_N, hipMemcpyDeviceToHost));

#elif OUTDEGREE
// do nothing for now

#else // not using Reduction, sum using hipblasSasum
    clock_gettime(CLOCK_REALTIME, &count_start);
    if (gb && (gb->gbExp[1].func == COUNT || gb->gbExp[0].func == COUNT)) {
        int h_gbCount = 0;
        cudaErrCheck(hipMemcpy(&h_gbCount, gbCount, sizeof(int), hipMemcpyDeviceToHost));
        printf("groupBy count: %d\n", h_gbCount);
        double gbCount_elapse = (gbCount_end.tv_sec -  gbCount_start.tv_sec)* BILLION + gbCount_end.tv_nsec - gbCount_start.tv_nsec;
        printf("GroupBy Time: %lf(ms)\n", gbCount_elapse/(1000*1000));

    } else {
        // previous calculate by hipblasHgemm: need conversion
        convertFp16ToFp32<<< (MATRIX_M * MATRIX_N + 255) / 256, 256 >>> (c_cublas, c_fp16_cublas, MATRIX_M * MATRIX_N);
    }

    uint64_t input_len = MATRIX_M*MATRIX_N;
    int asum_len = 200000000; // Sasum addition per section

    hipblasStatus_t ret;
    ret = hipblasCreate(&cublasHandle);
//    printf("input_len: %lu\n", input_len);

    if (input_len < asum_len) {
        float *cb_res = (float*)malloc(sizeof(float));
        ret = hipblasSasum(cublasHandle, MATRIX_M*MATRIX_N, c_cublas, 1, cb_res);
        clock_gettime(CLOCK_REALTIME, &count_end);
        printf("c_host_cublas sum: %.0f\n", *cb_res);
    } else { // support on machine has sufficient device memory ~15GB
        int num_sec = (int)(ceil(input_len/(float)asum_len));
        int remain = input_len % asum_len;
        float cb_res = 0;
        uint64_t pos = 0;
        uint64_t sum_res = 0;
        int i;
        for (i = 0; i < num_sec-1; i++) {
            ret = hipblasSasum(cublasHandle, asum_len, c_cublas+pos, 1, &cb_res);
            pos += asum_len;
            sum_res += (uint64_t)cb_res;
            //printf("i: %d\tcb_res: %f\tsum_res: %lu\n",i,cb_res,sum_res);
        }
        ret = hipblasSasum(cublasHandle, remain, c_cublas+pos, 1, &cb_res);
        sum_res += (uint64_t)cb_res;
        clock_gettime(CLOCK_REALTIME, &count_end);
        printf("c_host_cublas sum: %lu\n", sum_res);
    }
#endif
    printf("cublasEX tensor cores (FP16) took %fms\n", cublasEXTime);
    
    free(c_host_cublas);
//    cudaErrCheck(hipFree(c_cublas));
#elif CUBLAS
    float cublasTime;

//    cudaErrCheck(hipMemcpy(c_host_sgemm, c_sgemm, sizeof(float)*MATRIX_M*MATRIX_N, hipMemcpyDeviceToHost));

    cudaErrCheck(hipEventSynchronize(stopcublas));
    cudaErrCheck(hipEventElapsedTime(&cublasTime, startcublas, stopcublas));
    clock_gettime(CLOCK_REALTIME, &count_start);
    /*
    hipblasStatus_t sgemm_ret;
    sgemm_ret = hipblasCreate(&cublasHandle_default);
    float *cbsgemm_res = (float*)malloc(sizeof(float));
    sgemm_ret = hipblasSasum(cublasHandle_default, MATRIX_M*MATRIX_N, c_sgemm, 1, cbsgemm_res);
    clock_gettime(CLOCK_REALTIME, &count_end);
    printf("c_host_sgemm sum: %.0f\n", *cbsgemm_res);
    */
    printf("cublas sgemm (FP32) took %fms\n", cublasTime);

    cudaErrCheck(hipEventDestroy(startcublas));
    cudaErrCheck(hipEventDestroy(stopcublas));
    free(c_host_sgemm);
    cudaErrCheck(hipFree(c_sgemm));

#endif

// free those data structures
#ifdef CUBLAS_HALF


#elif CUBLAS
    free(h_fp32_A);
    free(h_fp32_B);
#endif

    clock_gettime(CLOCK_REALTIME, &tcu_end);
    double tcu_fill = (fill_end.tv_sec -  fill_start.tv_sec)* BILLION + fill_end.tv_nsec - fill_start.tv_nsec;
    //double tcu_convert = (convert_end.tv_sec -  convert_start.tv_sec)* BILLION + convert_end.tv_nsec - convert_start.tv_nsec;
    double tcu_elapse = (tcu_end.tv_sec -  tcu_start.tv_sec)* BILLION + tcu_end.tv_nsec - tcu_start.tv_nsec;
    double init_elapse = (init_end.tv_sec -  init_start.tv_sec)* BILLION + init_end.tv_nsec - init_start.tv_nsec;
    double cuMemcpy_elapse = (cuMemcpy_end.tv_sec -  cuMemcpy_start.tv_sec)* BILLION + cuMemcpy_end.tv_nsec - cuMemcpy_start.tv_nsec;
#if defined(CUBLAS_HALF) || defined(CUBLAS)
    double count_elapse = (count_end.tv_sec -  count_start.tv_sec)* BILLION + count_end.tv_nsec - count_start.tv_nsec;
    //double debug_elapse = (debug_end.tv_sec -  debug_start.tv_sec)* BILLION + debug_end.tv_nsec - debug_start.tv_nsec;
    //double transpose_elapse = (transpose_end.tv_sec -  transpose_start.tv_sec)* BILLION + transpose_end.tv_nsec - transpose_start.tv_nsec;
#endif


#ifdef PAGERANK
    double pagerankVerify_elapse = (pagerankVerify_end.tv_sec -  pagerankVerify_start.tv_sec)* BILLION + pagerankVerify_end.tv_nsec - pagerankVerify_start.tv_nsec;
#endif
    
    printf("Initialization: %lf(ms)\n", init_elapse/(1000*1000));
    printf("Matrices filling: %lf(ms)\n", tcu_fill/(1000*1000));
    printf("hipMemcpy: %lf(ms)\n", cuMemcpy_elapse/(1000*1000));
    printf("MMA total time: %lf(ms)\n", tcu_elapse/(1000*1000));
#ifdef CUBLAS_HALF

if (gb && (gb->gbExp[1].func == COUNT || gb->gbExp[0].func == COUNT)) {
    //printf("cublasEX join time: %lf(ms)\n", test_elapse/(1000*1000));
    printf("cublasEX sum counting: %lf(ms)\n", count_elapse/(1000*1000));
}
    //printf("hipblasCreate cold start: %lf(ms)\n", debug_elapse/(1000*1000));
    //printf("gpu transpose: %lf(ms)\n", transpose_elapse/(1000*1000));
#ifdef PAGERANK
    printf("PageRank Verify hipMemcpy time: %lf(ms)\n", pagerankVerify_elapse/(1000*1000));
#endif

#elif CUBLAS
    //printf("cublasSGEMM sum counting: %lf(ms)\n", count_elapse/(1000*1000));
    printf("hipblasCreate cold start: %lf(ms)\n", debug_elapse/(1000*1000));
    //printf("gpu transpose: %lf(ms)\n", transpose_elapse/(1000*1000));
#endif
//#ifdef DEBUG
    //cudaPrintfDisplay(stdout, true);
    //cudaPrintfEnd();
//#endif
    return res; // FIXME: return res table if second join need this as input  

}
