#include "hip/hip_runtime.h"
/*
   Copyright (c) 2012-2013 The Ohio State University.

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.
*/


#include <stdio.h>
#include <stdlib.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <sys/fcntl.h>
#include <sys/mman.h>
#include <unistd.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include "../include/common.h"
#include "../include/tcuJoin.h"
#include "../include/gpuCudaLib.h"
#include "scanImpl.cu"
#include <hip/hip_fp16.h>
#include <hiprand.h>
#include <mma.h>
#include <hipblas.h>
#include <math.h>
#ifdef DEBUG
#include "../include/cuPrintf.cu"
#include "../include/cuPrintf.cuh"
#endif

using namespace nvcuda;

// For wmma API, these must be multiples fo 16
//#define MATRIX_M 16
//#define MATRIX_N 16
//#define MATRIX_K 16

const int WMMA_M = 16;
const int WMMA_N = 16;
const int WMMA_K = 16;

// Define some error checking macros.
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
    if (stat != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
    }
}

/*
#define curandErrCheck(stat) { curandErrCheck_((stat), __FILE__, __LINE__); }
void curandErrCheck_(hiprandStatus_t stat, const char *file, int line) {
    if (stat != HIPRAND_STATUS_SUCCESS) {
        fprintf(stderr, "cuRand Error: %d %s %d\n", stat, file, line);
    }
}

#define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
    }
}
*/

__host__ void static verify_result(float * matrix, int height, int width) {
    int i;
    for (i = 0; i < height*width; i++) {
    //for (i = width*15; i < width*16; i++) {
        //printf("%d\t", matrix[i]);
        printf("%.0f\t", matrix[i]);
        if ((i+1) % width == 0)
            printf("\n\n");  
    }

}

/* Transpose the matrix on CPU */
__host__ void transpose(float *in, float *out, int row, int col) {
    for (int i = 0; i < row; i++) {
        for (int j = 0; j < col; j++) {
            out[j*row+i] = in[i*col+j];
        }
    }
}

__host__ int sum_matrix(float *mat, int height, int width) {
    int i, sum = 0;
    for (i = 0; i < height*width; i++)
        sum += mat[i];
    return sum;
}

/* Find the nearest multiple of N, check the width of matrix or tupleNum to form the matrices for MM */
__host__ int nearestMultipleN(int inNum, int n) {
    if (!n)
        return inNum;
    int remain = inNum % n;
    if (!remain)
        return inNum;
    return (inNum + n - remain);
}

/*
 *  If the query only need to return the count of join result.
 *  result t = mat1*mat.T
 *  count = t.size - sum(t) -- how many non-zero in t
 */
__host__ void static tcu_match(struct joinNode *jNode, int width,
         float *A, float *B, int attr_type1, int attr_type2) {

    int A_tupleNum = jNode->leftTable->tupleNum;
    int B_tupleNum = jNode->rightTable->tupleNum;

    // create first matrix
    int i, colContIdx; // index of column content
    colContIdx = 0;
    for (i = 0; i < A_tupleNum; i++) {
        int *colCont;   // content of column
        colCont = (int*)(&jNode->leftTable->content[jNode->leftKeyIndex][colContIdx]);
        colContIdx += attr_type1; // 4 because of INT type
        A[i*width+(*colCont)] = 1; // mark as 1 if appear in the matrix
    }

    // create second matrix
    colContIdx = 0;
    for (i = 0; i < B_tupleNum; i++) {
        int *colCont;
        colCont = (int*)(&jNode->rightTable->content[jNode->rightKeyIndex][colContIdx]);
        colContIdx += attr_type1;
        B[i*width+(*colCont)] = 1;
    }

    // transpose second matrix
    //transpose(B, B_T, B_tupleNum, width);

    // perform MM & return count on device
}

/* Map the table entires into matrix for tensor core to use 
 * Assum both matrix have the same dimension and the value in INT type for now, e.g., both 16x16 dim
 * To support multiple types, this function need to be modified
 */

// micro benchmark for simple matrix multiplication query
__host__ void static micro_mm(struct joinNode *jNode, float * matrix1, float * matrix2, int width,
        int attr_num1, int attr_num2, int attr_type1, int attr_type2) {
    int *mat1_i, *mat1_j, *mat1_val; // row index, col index, value
    int *mat2_i, *mat2_j, *mat2_val;

    int leftTupleNum = jNode->leftTable->tupleNum;
    int rightTupleNum = jNode->rightTable->tupleNum;
 
    mat1_i = (int*)malloc(sizeof(int) * leftTupleNum); 
    mat1_j = (int*)malloc(sizeof(int) * leftTupleNum); 
    mat1_val = (int*)malloc(sizeof(int) * leftTupleNum); 
   
    mat2_i = (int*)malloc(sizeof(int) * rightTupleNum); 
    mat2_j = (int*)malloc(sizeof(int) * rightTupleNum); 
    mat2_val = (int*)malloc(sizeof(int) * rightTupleNum); 

    int i, j; 
    for (i = 0; i < attr_num1; i++) {
        int left_col_idx = jNode->leftTable->attrIndex[i];
        int k = 0; // k is row-index of the table (tupleNum index)
        
        for (j = 0; j < leftTupleNum * attr_type1; j+=attr_type1) {
            int *temp;
            temp = (int*)(&jNode->leftTable->content[i][j]);
            
            if (left_col_idx == 0) { // match to schema's i
                mat1_i[k] = *temp;
            }
            else if (left_col_idx == 1) { // match to schema's j
                mat1_j[k] = *temp;
            }
            else { // match to schema's val
                // read 4 bytes at once because the type is int
                mat1_val[k] = *temp;
            }
            k++;
        }
    }

    
    for (i = 0; i < attr_num2; i++) {
        int right_col_idx = jNode->rightTable->attrIndex[i];
        int k = 0;
        
        for (j = 0; j < rightTupleNum * attr_type2; j+=attr_type2) {
            int *temp;
            temp = (int*)(&jNode->rightTable->content[i][j]);
            
            if (right_col_idx == 0) {
                mat2_i[k] = *temp;
            }
            else if (right_col_idx == 1) {
                mat2_j[k] = *temp;
            }
            else {
                mat2_val[k] = *temp;
            }
            k++;
        }
    }

    // map index to array[width * i + j] = val
    // prepare two matrices (1-D array format) for WMMA
    int m;
    for (m = 0; m < leftTupleNum; m++) {
        matrix1[width * mat1_i[m] + mat1_j[m]] = (float)mat1_val[m];
        //printf("%.2f\t", matrix1[width * mat1_i[m] + mat1_j[m]]);
    }
    //printf("\n");

    //printf("rightTupleNum: %d\n", rightTupleNum);
    for (m = 0; m < rightTupleNum; m++) {
        //printf("mat2 val: %d\tn: %d\n", mat2_val[n], n);
        matrix2[width * mat2_i[m] + mat2_j[m]] = (float)mat2_val[m];
        //printf("%.2f\t", matrix2[width * mat2_i[m] + mat2_j[m]]);
    }
    //printf("\n");

    free(mat1_i);
    free(mat1_j);
    free(mat1_val);
    free(mat2_i);
    free(mat2_j);
    free(mat2_val);
}

/* Print matrix content in device memory */
#ifdef DEBUG
__global__ void static verify_gpuResult(half * matrix, int width) {
    int i;
    for (i = 0; i < width*width; i++) {
        //printf("%d\t", matrix[i]);
        cuPrintf("%.1f\t", __half2float(matrix[i]));
        if ((i+1) % width == 0)
          cuPrintf("\n");  
    }

}
#endif

/* Convert input data from half to float type */
__global__ void static convertFp16ToFp32(float *out, half *in, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        out[idx] = __half2float(in[idx]);
        /*
        if (out[idx])  {
            cuPrintf("idx: %d\t%.1f\n", idx, __half2float(out[idx])); 
        }
        */
        //out[idx] = in[idx];
    }
}

/* Convert matrix from int to half type */
__global__ void static convertFp32ToFp16(half *out, float *in, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        out[idx] = __float2half(in[idx]);
        /*
        if (out[idx])  {
            cuPrintf("idx: %d\t%.1f\n", idx, __half2float(out[idx])); 
        }
        */
        //out[idx] = in[idx];
    }
}


/* Convert matrix from int to float type */
__host__ void static convertIntToFp32(float *out, int *in, int width) {
    int i;
    for (i = 0; i < width * width; i++) {
        out[i] = (float)in[i]; 
    }
}

/* Check whether the tupleNum is multiple of 16 because the WMMA requires the width of matrix be multiple of 16 */
__host__ int static findMatWidth(int tupleNum) {
    if (tupleNum <= 256)
        return 16;
    else {
        int tmp = ceil(sqrt(tupleNum));
        return (int)(ceil(tmp/(float)16)*16);
    }
}

__device__ static float getVal(char **content, struct mathExp exp, int pos) {
    float res;
    if (exp.opType == CONS)
        res = exp.opValue;
    else {
        int index = exp.opValue;
        res = ((int *)(content[index]))[pos];
    }

    return res;
}

// since WMMA perform C = alpha*A*B+beta*C, here we just fill operator MULTIPLY
__device__ static void fillMathExp(char **content, struct mathExp exp, int pos, float * A, float * B) {

    if (exp.op == MULTIPLY) {
        if (((struct mathExp*)exp.exp)[0].op == NOOP)
            A[pos] = getVal(content, ((struct mathExp*)exp.exp)[0], pos);
        if (((struct mathExp*)exp.exp)[1].op == NOOP)
            B[pos] = getVal(content, ((struct mathExp*)exp.exp)[1], pos);
    }
        
    return;
}

/* set the first column of the matrix to be 1.0 */
__host__ static void set_mask(float *mask, int height, int width) {
    for (int i = 0; i < height*width; i+=width) {
        mask[i] = 1.0;
    }
}

/* set the first row of the matrix to be 1.0 */
__host__ static void set_mask2(float *mask, int height, int width) {
    
    for (int i = 0; i < width; i++) {
        mask[i] = 1.0;
    }
    
}

__global__ static void agg_cal_cons(char ** content, int colNum, struct groupByExp* exp, long tupleNum, float * A, float * B) {
    int stride = blockDim.x * gridDim.x;
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    for(int i=index;i<tupleNum;i+=stride){
        for(int j=0;j<colNum;j++){
            int func = exp[j].func;
            // for now, we only care about SUM
            if (func == SUM) {
                // 1. fill data into two matrices
                //transform_data(content, exp[j].exp, i, A, B);
                fillMathExp(content, exp[j].exp, i, A, B);
                // TODO: how to maintain the order for threads
                // maybe the order does not important if we can get relative ranking

                // 2. copy data into device using hipMemcpy (if directly assign in device memory, can avoid this step)

            } else if (func == AVG) {
                // not the main point now
            }
        }
    }
} 

/* Performs an MxNxK GEMM (C=alpha*A*B + beta*C) assuming:
 *  1) Matrices are packed in memory.
 *  2) M, N and K are multiples of 16.
 *  3) Neither A nor B are transposed.
 */
__global__ void wmma_example(half *a, half *b, float *c, int M, int N, int K, float alpha, float beta) {
    // Leading dimensions. Packed with no transpositions.
    int lda = M;
    int ldb = K;
    int ldc = M;

    // Tile using a 2D grid
    int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / warpSize;
    int warpN = (blockIdx.y * blockDim.y + threadIdx.y);

    // Declare the fragments
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> b_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> acc_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;

    wmma::fill_fragment(acc_frag, 0.0f);

    // Loop over k
    for (int i = 0; i < K; i += WMMA_K) {
        int aRow = warpM * WMMA_M;
        int aCol = i;

        int bRow = i;
        int bCol = warpN * WMMA_N;

        // Bounds checking
        if (aRow < M && aCol < K && bRow < K && bCol < N) { 
            // Load the inputs
            wmma::load_matrix_sync(a_frag, a + aRow + aCol * lda, lda);
            wmma::load_matrix_sync(b_frag, b + bRow + bCol * ldb, ldb);

            // Perform the matrix multiplication
            wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
        }

    }

    // Load in the current value of c, scale it by beta, and add this our result scaled by alpha
    int cRow = warpM * WMMA_M;
    int cCol = warpN * WMMA_N;

    if (cRow < M && cCol < N) {
        wmma::load_matrix_sync(c_frag, c + cRow + cCol * ldc, ldc, wmma::mem_row_major);

        for(int i=0; i < c_frag.num_elements; i++) {
            c_frag.x[i] = alpha * acc_frag.x[i] + beta * c_frag.x[i];
        }

        // Store the output
        wmma::store_matrix_sync(c + cRow + cCol * ldc, c_frag, ldc, wmma::mem_row_major);
    }
}

/*
 * tcuJoinn using NVIDIA's WMMA lib to perform matrix multiplication can aggregation..
 *
 * Prerequisites:
 *  1. the data to be joined can be fit into GPU device memory.
 *  2. dimension table is not compressed
 *  
 * Input:
 *  jNode: contains information about the two joined tables.
 *  pp: records statistics such as kernel execution time
 *
 * Output:
 *  A new table node
 */
struct tableNode * tcuJoin(struct joinNode *jNode, struct statistic *pp, int *matrix_dim){
#ifdef DEBUG
    cudaPrintfInit();
#endif
    int leftTupleNum = jNode->leftTable->tupleNum;
    int rightTupleNum = jNode->rightTable->tupleNum;

    // parse user input dimension from command line
    int MATRIX_M, MATRIX_N, MATRIX_K;
    MATRIX_M = nearestMultipleN(leftTupleNum, 16);
    MATRIX_N = nearestMultipleN(rightTupleNum, 16);
    // TODO: for CUBLAS_HALF, MATRIX_K should be other values
    MATRIX_K = *matrix_dim; // user input, matrix width
#ifdef DEBUG
    printf("left  tuple #: %d\n", leftTupleNum);
    printf("right tuple #: %d\n", rightTupleNum);
    printf("MATRIX_M: %d\n", MATRIX_M);
    printf("MATRIX_N: %d\n", MATRIX_N);
    printf("MATRIX_K: %d\n", MATRIX_K);
#endif

    struct timespec tcu_start, tcu_end;
    struct timespec init_start, init_end;
    struct timespec fill_start, fill_end;
    struct timespec convert_start, convert_end;
    struct timespec cuMemcpy_start, cuMemcpy_end;
    clock_gettime(CLOCK_REALTIME, &tcu_start);
    clock_gettime(CLOCK_REALTIME, &init_start);

#ifdef WMMA_INT4
    unsigned char *h_int_A, *h_int_B; // host int4 array
    unsigned char *d_int_A, *d_int_B; // device int4 array
    unsigned char *c_int_wmma, *c_host_int_wmma;
    int alpha = 1;
    int beta = 0;
#else
    float *h_fp32_A, *h_fp32_B; // host float32 array
    float *d_fp32_A, *d_fp32_B; // device float32 array
    half *d_fp16_A, *d_fp16_B;
    float *c_wmma, *c_wmma_sum1, *c_wmma_sum2, *c_host_wmma;
    float *d_fp32_mask, *h_fp32_mask;
    float *d_fp32_mask2, *h_fp32_mask2;
    half *d_fp16_mask;
    half *d_fp16_mask2;
    float alpha = 1.0f;
    float beta = 0.0f;
#endif

#ifdef CUBLAS_HALF
    float *c_cublas, *c_host_cublas;
    hiprandGenerator_t gen;
    // use tensor core or cublas
    hipblasHandle_t cublasHandle; // cublas tcu
    hipEvent_t startcublasEX;
    hipEvent_t stopcublasEX;

    cublasErrCheck(hipblasCreate(&cublasHandle));
    // enable tensor core
    cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH));
#elif CUBLAS
    float *c_sgemm, *c_host_sgemm;
    hiprandGenerator_t gen;
    hipblasHandle_t cublasHandle_default; // cublas default
    hipEvent_t startcublas; // for sgemm (FP32)
    hipEvent_t stopcublas;

    cublasErrCheck(hipblasCreate(&cublasHandle_default));
    cublasErrCheck(hipblasSetMathMode(cublasHandle_default, HIPBLAS_DEFAULT_MATH));
#else
    hipEvent_t startWMMA;
    hipEvent_t stopWMMA;
    cudaErrCheck(hipEventCreate(&startWMMA));
    cudaErrCheck(hipEventCreate(&stopWMMA)); 

    dim3 gridDim;
    dim3 blockDim;
    // blockDim.x must be a multple of warpSize
    // 128x4 means we have 16 warps and a block computes a 64x64 output tile
    blockDim.x = 128;
    blockDim.y = 4;

    gridDim.x = (MATRIX_M + (WMMA_M * blockDim.x / 32 - 1)) / (WMMA_M * blockDim.x / 32);
    gridDim.y = (MATRIX_N + WMMA_N * blockDim.y - 1) / (WMMA_N * blockDim.y);
#endif

#ifdef WMMA_INT4
    c_host_int_wmma = (int*)calloc(MATRIX_M*MATRIX_N, sizeof(int));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&c_int_wmma, MATRIX_M * MATRIX_N * sizeof(int)));
#elif CUBLAS_HALF
    c_host_cublas = (float*)calloc(MATRIX_M*MATRIX_N, sizeof(float));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&c_cublas, MATRIX_M * MATRIX_N * sizeof(float)));
#elif CUBLAS
    c_host_sgemm =  (float*)calloc(MATRIX_M*MATRIX_N, sizeof(float));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&c_sgemm, MATRIX_M * MATRIX_N * sizeof(float)));
#else
    c_host_wmma = (float*)calloc(MATRIX_M*MATRIX_N, sizeof(float));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&c_wmma, MATRIX_M * MATRIX_N * sizeof(float)));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&c_wmma_sum1, MATRIX_M * MATRIX_N * sizeof(float)));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&c_wmma_sum2, MATRIX_M * MATRIX_N * sizeof(float)));
#endif
   
#ifdef WMMA_INT4
    h_int_A = (unsigned char*)calloc(MATRIX_M*MATRIX_K, sizeof(unsigned char));
    h_int_B = (unsigned char*)calloc(MATRIX_K*MATRIX_N, sizeof(unsigned char));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&d_int_A, MATRIX_M * MATRIX_K * sizeof(unsigned char)));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&d_int_B, MATRIX_K * MATRIX_N * sizeof(unsigned char)));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&c_int_wmma, MATRIX_M * MATRIX_N * sizeof(int)));

#else
    h_fp32_A = (float*)calloc(MATRIX_M*MATRIX_K, sizeof(float));
    h_fp32_B = (float*)calloc(MATRIX_K*MATRIX_N, sizeof(float));
    h_fp32_mask = (float*)calloc(MATRIX_M*MATRIX_N, sizeof(float));
    h_fp32_mask2 = (float*)calloc(MATRIX_M*MATRIX_N, sizeof(float));

    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&d_fp32_A, MATRIX_M * MATRIX_K * sizeof(float)));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&d_fp32_B, MATRIX_K * MATRIX_N * sizeof(float)));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&d_fp32_mask, MATRIX_M * MATRIX_N * sizeof(float)));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&d_fp32_mask2, MATRIX_M * MATRIX_N * sizeof(float)));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&d_fp16_A, MATRIX_M * MATRIX_K * sizeof(half)));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&d_fp16_B, MATRIX_K * MATRIX_N * sizeof(half)));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&d_fp16_mask, MATRIX_M * MATRIX_N * sizeof(half)));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&d_fp16_mask2, MATRIX_M * MATRIX_N * sizeof(half)));

    set_mask(h_fp32_mask, MATRIX_M, MATRIX_N);
    set_mask2(h_fp32_mask2, MATRIX_M, MATRIX_N);

    //printf("mask2:\n");
    //verify_result(h_fp32_mask2, MATRIX_M, MATRIX_K);
#endif    

    clock_gettime(CLOCK_REALTIME, &init_end);

    clock_gettime(CLOCK_REALTIME, &fill_start); 
#ifdef WMMA_INT4

#elif WMMA_HALF    
    tcu_match(jNode, MATRIX_K, h_fp32_A, h_fp32_B, jNode->leftTable->attrType[0], jNode->rightTable->attrType[0]);
    /*    
    micro_mm(jNode, h_fp32_A, h_fp32_B, MATRIX_M,
            jNode->leftTable->totalAttr, jNode->rightTable->totalAttr, jNode->leftTable->attrType[0], jNode->rightTable->attrType[0]);

    */
#else
#endif
    clock_gettime(CLOCK_REALTIME, &fill_end);

    clock_gettime(CLOCK_REALTIME, &cuMemcpy_start);
#ifdef WMMA_INT4
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(d_int_A, h_int_A, sizeof(unsigned char) * MATRIX_M * MATRIX_K, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(d_int_B, h_int_B, sizeof(unsigned char) * MATRIX_K * MATRIX_N, hipMemcpyHostToDevice));
#else
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(d_fp32_A, h_fp32_A, sizeof(float) * MATRIX_M * MATRIX_K, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(d_fp32_B, h_fp32_B, sizeof(float) * MATRIX_K * MATRIX_N, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(d_fp32_mask, h_fp32_mask, sizeof(float) * MATRIX_M * MATRIX_N, hipMemcpyHostToDevice));
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(d_fp32_mask2, h_fp32_mask2, sizeof(float) * MATRIX_M * MATRIX_N, hipMemcpyHostToDevice));
#endif
    clock_gettime(CLOCK_REALTIME, &cuMemcpy_end);

    clock_gettime(CLOCK_REALTIME, &convert_start); // if float->half
#ifdef WMMA_INT4
#else
    convertFp32ToFp16<<< (MATRIX_M * MATRIX_K + 255) / 256, 256 >>> (d_fp16_A, d_fp32_A, MATRIX_M * MATRIX_K);
    convertFp32ToFp16<<< (MATRIX_K * MATRIX_N + 255) / 256, 256 >>> (d_fp16_B, d_fp32_B, MATRIX_K * MATRIX_N);
    convertFp32ToFp16<<< (MATRIX_N * MATRIX_K + 255) / 256, 256 >>> (d_fp16_mask, d_fp32_mask, MATRIX_M * MATRIX_N);
    convertFp32ToFp16<<< (MATRIX_N * MATRIX_K + 255) / 256, 256 >>> (d_fp16_mask2, d_fp32_mask2, MATRIX_M * MATRIX_N);
#endif
    clock_gettime(CLOCK_REALTIME, &convert_end);
#ifdef WMMA_HALF 
    printf("\nM = %d, N = %d, K = %d. alpha = %f, beta = %f\n\n", MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta);

    printf("Running with wmma...\n");
    cudaErrCheck(hipEventRecord(startWMMA));
    wmma_example <<< gridDim, blockDim >>> (d_fp16_A, d_fp16_B, c_wmma, MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta); 
    
    half *c_wmma_reduction1, *c_wmma_reduction2;
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&c_wmma_reduction1, MATRIX_M * MATRIX_N * sizeof(half)));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&c_wmma_reduction2, MATRIX_M * MATRIX_N * sizeof(half)));
    convertFp32ToFp16<<< (MATRIX_M * MATRIX_N + 255) / 256, 256 >>> (c_wmma_reduction1, c_wmma, MATRIX_M * MATRIX_N);

    wmma_example <<< gridDim, blockDim >>> (d_fp16_mask2, c_wmma_reduction1, c_wmma_sum1, MATRIX_M, MATRIX_N, MATRIX_M, alpha, beta); 
    convertFp32ToFp16<<< (MATRIX_M * MATRIX_N + 255) / 256, 256 >>> (c_wmma_reduction2, c_wmma_sum1, MATRIX_M * MATRIX_N);

    wmma_example <<< gridDim, blockDim >>> (d_fp16_mask2, c_wmma_reduction2, c_wmma_sum2, MATRIX_M, MATRIX_N, MATRIX_M, alpha, beta); 
    
    cudaErrCheck(hipEventRecord(stopWMMA));
#elif CUBLAS_HALF
    printf("Running with cuBLAS on TCUs...\n");
    cudaErrCheck(hipEventRecord(startcublasEX));
    cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                MATRIX_M, MATRIX_N, MATRIX_K,
                &alpha,
                d_fp16_A, HIP_R_16F, MATRIX_M,
                d_fp16_B, HIP_R_16F, MATRIX_N,
                &beta,
                c_cublas, HIP_R_32F, MATRIX_K,
                HIP_R_32F, CUBLAS_GEMM_DFALT_TENSOR_OP)); // tcu
    cudaErrCheck(hipEventRecord(stopcublasEX));
#elif CUBLAS
    printf("Running with sgemm...\n");
    cudaErrCheck(hipEventRecord(startcublas));
    hipblasSgemm(cublasHandle_default, HIPBLAS_OP_N, HIPBLAS_OP_N, MATRIX_M, MATRIX_N, MATRIX_K, &alpha, d_fp32_A, MATRIX_M, d_fp32_B, MATRIX_N, &beta, c_sgemm, MATRIX_K);
    cudaErrCheck(hipEventRecord(stopcublas));
#endif    

#ifdef WMMA_HALF
    struct timespec tmp_start, tmp_end;
    clock_gettime(CLOCK_REALTIME, &tmp_start);
    cudaErrCheck(hipMemcpy(c_host_wmma, c_wmma_sum2, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToHost));

    //printf("c_host_wmma:\n");
    //verify_result(c_host_wmma, MATRIX_M, MATRIX_N);

    printf("Number of join results (MM reduction): %.0f\n", c_host_wmma[0]);
    printf("Number of join results (CPU count): %d\n", sum_matrix(c_host_wmma, MATRIX_M, MATRIX_N));
    clock_gettime(CLOCK_REALTIME, &tmp_end);
#elif CUBLAS_HALF
    cudaErrCheck(hipMemcpy(c_host_cublas, c_cublas, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToHost));
#elif CUBLAS
    cudaErrCheck(hipMemcpy(c_host_sgemm, c_sgemm, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToHost));
#endif

    // print error checking, hipblasGemmEx and cublas
    //printf("\nChecking results with tensor cores...\n");

    // 0.01% relative tolerance. 1e-5 absolute tolerance.
    /*
    int errors = 0;
    for (int i = 0; i < MATRIX_M * MATRIX_N; i++) {
        float v1 = c_host_sgemm[i];
        float v2 = c_host_cublas[i];

        // TODO: abs diff failed due to precision loss
        // current fix: range value less than 2^10 (IEEE half type)
        if (v1 / v2 > 1.0001 || v2 / v1 > 1.0001 || abs(v1 - v2) > 1e-3) {
            errors++;
            if (errors < 10) printf("%.1f %.1f diff:%.1f\n", v1, v2, abs(v1-v2));
        }
    }

    if (errors > 0) {
        printf("WMMA does not agree with cuBLAS! %d errors!\n", errors);
    }
    */
#ifdef CUBLAS_HALF
    float cublasEXTime;

    cudaErrCheck(hipEventSynchronize(stopcublasEX));
    cudaErrCheck(hipEventElapsedTime(&cublasEXTime, startcublasEX, stopcublasEX));
    printf("cublasEX tensor cores (FP16) took %fms\n", cublasEXTime);

    cudaErrCheck(hipEventDestroy(startcublasEX));
    cudaErrCheck(hipEventDestroy(stopcublasEX));
    free(c_host_cublas);
    cudaErrCheck(hipFree(c_cublas));
#elif CUBLAS
    float cublasTime;

    cudaErrCheck(hipEventSynchronize(stopcublas));
    cudaErrCheck(hipEventElapsedTime(&cublasTime, startcublas, stopcublas));
    printf("cublas sgemm (FP32) took %fms\n", cublasTime);

    cudaErrCheck(hipEventDestroy(startcublas));
    cudaErrCheck(hipEventDestroy(stopcublas));
    free(c_host_sgemm);
    cudaErrCheck(hipFree(c_sgemm));
#else
    float wmmaTime;

    cudaErrCheck(hipEventSynchronize(stopWMMA));
    cudaErrCheck(hipEventElapsedTime(&wmmaTime, startWMMA, stopWMMA));
    printf("wmma took %fms\n", wmmaTime);

    // free those data structures
    cudaErrCheck(hipEventDestroy(startWMMA));
    cudaErrCheck(hipEventDestroy(stopWMMA));
#endif

#ifdef WMMA_HALF
    free(c_host_wmma);
    cudaErrCheck(hipFree(c_wmma));
#endif

#ifdef WMMA_INT4
    free(h_int_A);
    free(h_int_B);
    free(c_host_int_wmma);
    cudaErrCheck(hipFree(d_int_A));
    cudaErrCheck(hipFree(d_int_B));
    cudaErrCheck(hipFree(c_int_wmma));
#else
    free(h_fp32_A);
    free(h_fp32_B);
    free(h_fp32_mask2);
    cudaErrCheck(hipFree(d_fp32_A));
    cudaErrCheck(hipFree(d_fp16_A));
    cudaErrCheck(hipFree(d_fp32_B));
    cudaErrCheck(hipFree(d_fp16_B));
    cudaErrCheck(hipFree(d_fp32_mask2));
    cudaErrCheck(hipFree(d_fp16_mask2));
#endif
    clock_gettime(CLOCK_REALTIME, &tcu_end);
    double tcu_fill = (fill_end.tv_sec -  fill_start.tv_sec)* BILLION + fill_end.tv_nsec - fill_start.tv_nsec;
    double tcu_convert = (convert_end.tv_sec -  convert_start.tv_sec)* BILLION + convert_end.tv_nsec - convert_start.tv_nsec;
    double tcu_elapse = (tcu_end.tv_sec -  tcu_start.tv_sec)* BILLION + tcu_end.tv_nsec - tcu_start.tv_nsec;
    double init_elapse = (init_end.tv_sec -  init_start.tv_sec)* BILLION + init_end.tv_nsec - init_start.tv_nsec;
    double cuMemcpy_elapse = (cuMemcpy_end.tv_sec -  cuMemcpy_start.tv_sec)* BILLION + cuMemcpy_end.tv_nsec - cuMemcpy_start.tv_nsec;
    double tmp_elapse = (tmp_end.tv_sec -  tmp_start.tv_sec)* BILLION + tmp_end.tv_nsec - tmp_start.tv_nsec;
    
    printf("Initialization: %lf(ms)\n", init_elapse/(1000*1000));
    printf("Matrices filling: %lf(ms)\n", tcu_fill/(1000*1000));
    printf("Data type convertion: %lf(ms)\n", tcu_convert/(1000*1000));
    printf("hipMemcpy: %lf(ms)\n", cuMemcpy_elapse/(1000*1000));
    printf("MMA end-to-end: %lf(ms)\n", tcu_elapse/(1000*1000));
    printf("Result verification: %lf(ms)\n", tmp_elapse/(1000*1000));
#ifdef DEBUG
    cudaPrintfDisplay(stdout, true);
    cudaPrintfEnd();
#endif
    return 0; // non-void function

}
