#include "hip/hip_runtime.h"
/*
   Copyright (c) 2012-2013 The Ohio State University.

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.
*/


#include <stdio.h>
#include <stdlib.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <sys/fcntl.h>
#include <sys/mman.h>
#include <unistd.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include "../include/common.h"
#include "../include/tcuJoin.h"
#include "../include/gpuCudaLib.h"
#include "scanImpl.cu"
#include <hip/hip_fp16.h>
#include <hiprand.h>
#include <mma.h>
#include <hipblas.h>
#include <math.h>
//#ifdef DEBUG
//#include "../include/cuPrintf.cu"
//#include "../include/cuPrintf.cuh"
//#endif

using namespace nvcuda;

#define MAX_THREADS 1024 // For NVIDIA Turing Architecture

// Define some error checking macros.
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
    if (stat != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s %s %d\n",
                hipGetErrorString(stat),
                file, line);
    }
}

#if defined(CUBLAS) || defined(CUBLAS_HALF)
#define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
    }
}
#endif

void* cublasCreateThread(void *x)
{
    hipblasHandle_t* cublasHandle = (hipblasHandle_t *)x;
    cublasErrCheck(hipblasCreate(cublasHandle));
    cublasErrCheck(hipblasSetMathMode(*cublasHandle,HIPBLAS_TENSOR_OP_MATH));
    return NULL;
}

__global__ static void count_op(float *red_sum, int length) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i > length) return;
    if (red_sum[i] != 0)
        return;
}

__global__ static void gb_count(float *red_sum, int length, int *cnt) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i >= length) return;
    if (red_sum[i] != 0)
        atomicAdd(cnt, 1);

}

/* Fill the actual float value for PageRank calculation. 
   Pagerank.ranking and Outdegree.degree */
__global__ void pagerank(char *columnIdx, char *columnVal, int matWidth, half *mat, size_t tupleNum, int attrTypeSize, int attrType, float pagerank_cons) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < tupleNum) {
        int stripe = i * attrTypeSize;
        int *id    = (int*)&columnIdx[stripe];

        if (attrType == INT) {
            int *val = (int*)&columnVal[stripe];
            mat[i*matWidth + (*id)] = __float2half((float)1/(*val));
            //cuPrintf("mat[%d]\t%d\n", i*matWidth + (*id), *val);
        } else if (attrType == FLOAT) {
            float *val   = (float*)&columnVal[stripe];
            
            mat[i*matWidth + (*id)] = __float2half((*val)*pagerank_cons);
            //cuPrintf("mat[%d]\t%.8f\n", i*matWidth + (*id), *val);
        }
    }
}

/* 
 *  Fill 1.0 on the index of unique value in the matrix;
 *  fill 0.0, otherwise. 
 */
__global__ void static gpu_fill(char *column, int matWidth, half *matA, 
        size_t tupleNum, int attrType) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= tupleNum) return;

    int index = i * attrType;
    //int value = (int)column[index]; // char -> int will lose 3 bytes
    int *value   = (int*)&column[index];
    matA[i*matWidth + (*value)] = __float2half(1.0f);
}

__global__ void static gpu_fill_2data(char *join_column, char *data_column, 
        char *data_column2, int matWidth_k, half *matA, size_t tupleNum, 
        int attrType, int scale) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= tupleNum) return;

    int index = i * attrType;
    int *join_value = (int*)&join_column[index];// col
    int *data_value = (int*)&data_column[index];// val
    int *data2 = (int*)&data_column2[index];    // row
    matA[(*data2) * matWidth_k + (*join_value)] = __float2half((float)(*data_value)/scale);
    //printf("matA[%d]: %f\n", (*data2) * matWidth_k + (*join_value), (float)(*data_value)/scale);
    //matA[(*data2) * matWidth_k + (*join_value)] = __float2half(65504.0f);
    //printf("row: %d\tcol: %d\tval: %d\n", *data2, *join_value, *data_value);
}

/* Fill matrix with data value. */
__global__ void static gpu_fill_data(char *join_column, char *data_column, 
        int matWidth_k, half *matA, size_t tupleNum, int attrType) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= tupleNum) return;

    int index = i * attrType;
    int *join_value = (int*)&join_column[index];
    int *data_value = (int*)&data_column[index];
    matA[i * matWidth_k + (*join_value)] = __float2half((float)(*data_value));
    //printf("matA[%d]: %.0f\n",i * matWidth_k + (*join_value),(float)(*data_value));
}

__global__ void static gpu_fill_gb(char *join_column, char *data_column, 
        int matWidth_k, half *matA, size_t tupleNum, int attrType) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= tupleNum) return;

    int index = i * attrType;
    int *join_value = (int*)&join_column[index];
    int *data_value = (int*)&data_column[index];
    matA[(*data_value) * matWidth_k + (*join_value)] = __float2half(1.0f);
}

__global__ void static gpu_fill_data_transpose(char *join_column, 
        char *data_column, int matWidth_n, half *matB, size_t tupleNum, 
        int attrType) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= tupleNum) return;

    int index = i * attrType;
    int *join_value = (int*)&join_column[index];
    int *data_value = (int*)&data_column[index];
    matB[(*join_value) * matWidth_n + i] = __float2half((float)(*data_value));
}

/* Fill matrix with ones according to groupBy column in transpose format. */
__global__ void static gpu_fill_gb_transpose(char *join_column, 
        char *data_column, int matWidth_n, half *matB, 
        size_t tupleNum, int attrType) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= tupleNum) return;

    int index = i * attrType;
    int *join_value = (int*)&join_column[index];
    int *data_value = (int*)&data_column[index];
    //if (*data_value > 1998 || *data_value < 1991)
    //    printf("%d\n", *data_value);
    matB[(*join_value) * matWidth_n + (*data_value)] = __float2half(1.0f);
    //printf("matB[%d]: %.0f\n",(*join_value) * matWidth_n + (*data_value), 1.0f);
}

/*
 * Fill ones matrix in transpose matrix format.
 */
__global__ void static gpu_fill_transpose(char *column, int matWidth, 
        half *matB, size_t tupleNum, int attrType) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= tupleNum) return;

    int index = i * attrType;
    int *value   = (int*)&column[index];
    int pos = (*value)*tupleNum+i;
    matB[pos] = __float2half(1.0f);
}

/* Fill matrix in dense format for matrix multiplication */
__global__ void static microbenchmark(char *mat_i, char *mat_j, char *mat_val, 
        int matWidth, half *mat, size_t tupleNum, int attrType) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i >= tupleNum) return;

    int index = i * attrType;
    int *row  = (int*)&mat_i[index]; 
    int *col  = (int*)&mat_j[index]; 
    int *val  = (int*)&mat_val[index];
    mat[(*row)*matWidth+(*col)] = __int2half_rn(*val);
}

__global__ void static microbenchmark_transpose(char *mat_i, char *mat_j, 
        char *mat_val, int matWidth, half *mat, size_t tupleNum, int attrType) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i >= tupleNum) return;

    int index = i * attrType;
    int *row  = (int*)&mat_i[index]; 
    int *col  = (int*)&mat_j[index]; 
    int *val  = (int*)&mat_val[index];
    mat[(*col)*matWidth+(*row)] = __int2half_rn(*val);
}

__global__ void static outdegree_fill(char *column_val, half *mat, 
        size_t tupleNum, int attrType) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i >= tupleNum) return;

    int index = i * attrType;
    //int *colIndex   = (int*)&column_idx[index];
    int *val        = (int*)&column_val[index];
    //printf("idx: %d\tval: %d\n", i*matWidth + (*colIndex), (*val));
    mat[(*val)] = __hadd(mat[(*val)], __int2half_rn(1));
}

#ifdef CUBLAS_HALF
__global__ void gpu_transpose(half *odata, const half *idata, int row, int col) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int x = index % col;
    int y = index / col;

    if (x < col && y < row) {
        odata[x*row + y] = idata[y*col + x];
    }
}
#elif CUBLAS
__global__ void gpu_transpose(float *odata, const float *idata, int row, int col) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int x = index % col;
    int y = index / col;

    if (x < col && y < row) {
        odata[x*row + y] = idata[y*col + x];
    }
}
#endif

__global__ void static pageRankAdd(float *mat, int n, float pageRankAlpha, 
        int numNodes) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        if (mat[idx] > 1e-6)
        //if (__hgt(mat[idx], __float2half(1e-6))) // precision loss
            mat[idx] += (float)(1-pageRankAlpha)/numNodes;
            //mat[idx] += __float2half((1-pageRankAlpha)/numNodes);
    }
}

/* Convert input data from half to float type */
__global__ void static convertFp16ToFp32(float *out, half *in, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        out[idx] = __half2float(in[idx]);
    }
}

/* Convert input data from half to float type */
__global__ void static convertFp32ToFp16(half *out, float *in, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        out[idx] = __float2half(in[idx]);
    }
}

/* Convert input data from char to half type */
__global__ void static convertCharToFp16(half *out, char *in, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        out[idx] = __int2half_rn((int)in[idx]);
    }
}

__global__ void groupByCount(float *data, int n, int *gbCount) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        if (data[idx] > 0.000001) {
        //if (data[idx] > 0.001) {
            atomicAdd(gbCount, 1);
        }
    }
}

__global__ void static naiveCount(float *res, int n, int *count) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        if (res[idx] > 0.000001) {
            //__syncthreads();
            atomicAdd(count, 1);
            //printf("res[%d]: %f\n", idx, res[idx]);
            //printf("count: %d\n", *count);
        }
    }
}

__global__ void placeCount(float *out, float *in, unsigned size)
{
    unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if (tid >= size) return;
    if (in[tid] > 0.00001)
        out[tid] = 1.0f;
    else
        out[tid] = 0.0f;
}

__host__ uint64_t getCount(float *in, uint64_t size)
{
    const uint64_t asumLen = 2e9;

    hipblasStatus_t ret;
    hipblasHandle_t cublasHandle;
    hipblasCreate(&cublasHandle);

    uint64_t iter = size / asumLen;
    uint64_t overflow = size % asumLen;
    uint64_t offset = 0;
    float partialRes;
    uint64_t sumRes = 0;

    for (uint64_t i = 0; i < iter; i++)
    {
        ret = hipblasSasum(cublasHandle, asumLen, in+offset, 1, &partialRes);
        offset += asumLen;
        sumRes += (uint64_t) partialRes;
    }

    // handle overflow
    if (overflow)
    {
        ret = hipblasSasum(cublasHandle, overflow, in+offset, 1, &partialRes);
        sumRes += (uint64_t) partialRes;
    }

    hipblasDestroy(cublasHandle);
    return sumRes;
} 

__global__ void reductionCount(int *out, float *in, unsigned size) 
{
    __shared__ int partialSum[256];
    unsigned int tid   = threadIdx.x;
    unsigned int start = 2 * blockIdx.x * blockDim.x;

    partialSum[tid] = 0;
    if (tid >= size) return;

    if (tid + start + blockDim.x < size) 
    {
        if (in[tid + start] > 0.000001) 
        {
            partialSum[tid]++;
        }
        if (in[tid + start + blockDim.x] > 0.000001) 
        {
            partialSum[tid]++;
        }
    }
    else 
    {
        if (in[tid + start] > 0.000001) 
        {
            partialSum[tid]++;
        }
    }
    
    __syncthreads();
    //if (partialSum[tid])
    //    printf("tid: %d\t partialSum: %d\n", tid, partialSum[tid]);
    for (unsigned int stride = blockDim.x/2; stride > 0; stride /= 2) {
        __syncthreads();
        if (tid < stride) { // reduce to left triangle
            partialSum[tid] += partialSum[tid + stride];
        }
    }

    if (tid == 0)
        out[blockIdx.x] = partialSum[0];
}

/* set the first column of the matrix to be 1.0 */
__host__ static void set_mask(float *mask, int height, int width) {
    for (int i = 0; i < height*width; i+=width) {
        mask[i] = 1.0;
    }
}

/* set the first row of the matrix to be 1.0 */
__host__ static void set_mask2(float *mask, int height, int width) {
    
    for (int i = 0; i < width; i++) {
        mask[i] = 1.0;
    }
    
}

__host__ static void setVector(float *vec, int n) {
    for (int i = 0; i < n; i++)
        vec[i] = 1.0;
}

__host__ static void setRed(short *red, int n) {
    for (int i = 0; i < n; i++)
        red[i] = (short)1;
}

/* only for small matrix result verification. */
__host__ static void print_matrix(float *mat, int rows, int cols) {
    for (int i = 0; i < rows * cols; i++) {
        printf("%.0f\t", mat[i]);
        if ((i+1) % cols == 0) {
            printf("\n");
        }
    }
}

/* Get column index from aggregate function for later data copy. */
__host__ static void getValIndex(struct joinNode *jNode, struct groupByNode *gb, 
        int *lValIndex, int *rValIndex, 
        int &lgbIndex, int &rgbIndex, 
        //int *ldataColIndex, int &rdataColIndex) {
        int &ldataColIndex, int &rdataColIndex,
        int &ldata2) {

    for (int i = 0; i < jNode->leftOutputAttrNum; i++) {
        for (int j = 0; j < gb->numFuncExpCol; j++) {
            // find index of aggFunc, e.g. SUM(X.column_name)
            if (jNode->leftPos[i] == gb->funcExpColIndex[j]) {
                lValIndex[i] = jNode->leftOutputIndex[i];

                if (ldataColIndex == -1) {
                    ldataColIndex = jNode->leftOutputIndex[i];
                    //printf("agg left ldataColIndex[%d]: %d\n", i,ldataColIndex);
                }
            }
            for (int k = 0; k < gb->groupByColNum; k++) {
                if (jNode->leftPos[i] == gb->groupByIndex[k])
                    lgbIndex = 1;
            }
        }
        // TODO: hard to know which colIdx belong to sum() as value or use as one dimension
        if (ldata2 == -1) {
            ldata2 = jNode->leftOutputIndex[i];
        }
        /*
        if (ldataColIndex[i] == -1) {
            ldataColIndex[i] = jNode->leftOutputIndex[i];
            printf("left ldataColIndex[%d]: %d\n", i,ldataColIndex[i]);
        }*/
    } 
    
    for (int i = 0; i < jNode->rightOutputAttrNum; i++) {
        for (int j = 0; j < gb->numFuncExpCol; j++) {
            //if (jNode->rightPos[i] == gb->funcExpColIndex[j]) {
            //    rValIndex[i] = jNode->rightOutputIndex[i];

                if (rdataColIndex == -1) {
                    rdataColIndex = jNode->rightOutputIndex[i];
                    //printf("right rdataColIndex: %d\n", rdataColIndex);
                }
            //}

            for (int k = 0; k < gb->groupByColNum; k++) {
                if (jNode->rightPos[i] == gb->groupByIndex[k])
                    rgbIndex = 1;
            }
        }
    } 
}

/* Match the first groupBy attribute, return 0 (left), 1 (right)*/
__host__ static int getGbLeftRight(struct joinNode *jNode, struct groupByNode *gb, int &gbConstant, int &gbLeftRight) {
    if (gbConstant == 1) return -1;
    
    for (int i = 0; i < jNode->leftOutputAttrNum; i++) {
        if (jNode->leftPos[i] == gb->groupByIndex[0]) {
            return 0;
        }
    } 
    
    for (int i = 0; i < jNode->rightOutputAttrNum; i++) {
        if (jNode->rightPos[i] == gb->groupByIndex[0]) {
            return 1;
        }
    } 
    return -1;
}

/* Mimic the max() in relational database. */
__host__ int getMaxVal(char *column, size_t tupleNum, int attrType) {
    int localMax = 0;

    for (int i = 0; i < tupleNum; i++) {
        int *val = (int*)&column[i*attrType];
        if (localMax < *val) {
            localMax = *val;
        }
    }
    return localMax;
}

__host__ int getMinVal(char *column, size_t tupleNum, int attrType) {
    int localMin = 0;

    for (int i = 0; i < tupleNum; i++) {
        int *val = (int*)&column[i*attrType];
        if (localMin > *val) {
            localMin = *val;
        }
    }
    return localMin;
}

/* Need to copy values to device */
__global__ void getMaxValGPU(char *column, size_t tupleNum, int attrType, int *maxVal) {
    __shared__ int sharedMax;

    if (threadIdx.x == 0) {
        sharedMax = 0;
    }
    __syncthreads();

    int localMax = 0;
    for (int i = threadIdx.x; i < tupleNum; i += blockDim.x) {
        int index = i * attrType;
        int *value   = (int*)&column[index];

        if (localMax < abs(*value)) {
            localMax = abs(*value);
        }
    }

    atomicMax(&sharedMax, localMax);
    __syncthreads();
    
    if (threadIdx.x == 0) {
        *maxVal = sharedMax;
    }
}

struct gpu_timer {
    gpu_timer() {
        hipEventCreate(&m_start);
        hipEventCreate(&m_end);
        hipEventRecord(m_start, 0);
    }

    float milliseconds_elapsed() {
        float elapsed_time;
        hipEventRecord(m_end, 0);
        hipEventSynchronize(m_end);
        hipEventElapsedTime(&elapsed_time, m_start, m_end);
        return elapsed_time;      
    }

    float seconds_elapsed() {
        return milliseconds_elapsed() / 1000.0;
    }

  protected:
    hipEvent_t m_start, m_end;
};

/* Read tableNode and convert into Coo matrix.
 * transpose -- 0: NON-TRANSPOSE, 1: TRANSPOSE 
 * fillOne   -- 0: fill data value, 1: fill 1 */
void mat2coo(int XtupleNum, char *XjoinKey, char *Xdata,
             int *cooRowInd, int *cooColInd, float *cooValues,
             int transpose, int fillOne)
{
    if (transpose)
    {
        for (int i = 0; i < XtupleNum; i++)
        {
            cooRowInd[i] = (int)XjoinKey[i*sizeof(int)];
            cooColInd[i] = i;
            if (fillOne) {
                cooValues[i] = 1.0f;
            } else {
                cooValues[i] = (float)Xdata[i*sizeof(float)];
            }
        }
    } else
    {
        for (int i = 0; i < XtupleNum; i++)
        {
            cooRowInd[i] = i;
            cooColInd[i] = (int)XjoinKey[i*sizeof(int)];
            if (fillOne) {
                cooValues[i] = 1.0f;
            } else {
                cooValues[i] = (float)Xdata[i*sizeof(float)];
            }
        }
    }
}

/* If has groupBy keyword, one matrix width will need to update.
 * Instead of using tupleNum, using Xdata as one dimension.  */
void mat2coo_gb(int XtupleNum, char *XjoinKey, char *Xdata,
                int *cooRowInd, int *cooColInd, float *cooValues,
                int transpose, int fillOne)
{
    if (transpose)
    {
        for (int i = 0; i < XtupleNum; i++)
        {
            cooRowInd[i] = (int)XjoinKey[i*sizeof(int)];
            cooColInd[i] = (int)Xdata[i*sizeof(int)];
            if (fillOne) {
                cooValues[i] = 1.0f;
            } else {
                cooValues[i] = (float)Xdata[i*sizeof(float)];
            }
        }
    } else
    {
        for (int i = 0; i < XtupleNum; i++)
        {
            cooRowInd[i] = (int)Xdata[i*sizeof(int)];
            cooColInd[i] = (int)XjoinKey[i*sizeof(int)];
            if (fillOne) {
                cooValues[i] = 1.0f;
            } else {
                cooValues[i] = (float)Xdata[i*sizeof(float)];
            }
        }
    }
}

/* Convert matrix format from Coo to Csr. */
void coo2csr(int X_num_rows, int Xnnz,
             int *X_cooRowInd, int *X_cooColInd, float *X_cooValues,
             int *csrOffsets, int *csrColumns, float *csrValues)
{
    // check how elements in each row
    int *num_elems_each_row = (int*)calloc(X_num_rows, sizeof(int));

    // count num_elems
    for (int i = 0; i < Xnnz; i++)
    {
        num_elems_each_row[X_cooRowInd[i]]++;
    }

    // prefix sum
    for (int i = 0; i < X_num_rows; i++)
    {
        csrOffsets[i+1] = num_elems_each_row[i] + csrOffsets[i];
    }

    for (int i = 0; i < Xnnz; i++)
    {
        num_elems_each_row[X_cooRowInd[i]]--;
        int r = X_cooRowInd[i];
        int offset = csrOffsets[r] + num_elems_each_row[X_cooRowInd[i]];
        csrColumns[offset] = X_cooColInd[i];
        csrValues[offset] = X_cooValues[i];
    }
}

/*
 * tcuJoinn using NVIDIA's cuBLAS lib to perform matrix multiplication and aggregation.
 *
 * Prerequisites:
 *  1. the data to be joined can be fit into GPU device memory.
 *  2. dimension table is not compressed
 *  3. user know the matrix dimension (#uniq values)
 *  
 * Input:
 *  jNode: contains information about the two joined tables.
 *  pp: records statistics such as kernel execution time
 *  matrix_dim: matrix width (number of unique values)
 *  gb: contains groupby information
 *
 * Output:
 *  Number of join counts and groupBy count if query contains groupBy keyword.
 *
 * Assumptions:
 *
 * 1. Two joined table schemas are the same for the simplicity of query parser.
 * 2. For all demo cases, all column types are INT, only PageRank queries 
 *    contain constant variable such as alpha and number of nodes.
 * 3. To support complex customized queries, code_gen.py modification is required.
 * 4. Metadata such as sparsity and number of non-zero elements are known from user.
 * Here, we have our assumption based on our filling methods.
 */
struct tableNode * tcuJoin(struct joinNode *jNode, struct statistic *pp, 
        int *matrix_dim, struct groupByNode *gb)
{

    //struct timespec tcu_start, tcu_end;
    //struct timespec init_start, init_end;
    //struct timespec cuMemcpy_start, cuMemcpy_end;
    //struct timespec fill_start, fill_end;
    //struct timespec count_start, count_end;
    float initTime, cudaMemcpyTime, fillTime, end2endTime;
    float tcu_compute_time, tcu_groupBy_time;

    struct tableNode * res = NULL;
    int leftTupleNum  = jNode->leftTable->tupleNum;
    int rightTupleNum = jNode->rightTable->tupleNum;
    uint64_t MATRIX_M, MATRIX_N, MATRIX_K; // avoid overflow
    uint64_t Annz, Bnnz;

    res = (struct tableNode*) malloc(sizeof(struct tableNode));
    CHECK_POINTER(res);
    res->totalAttr = jNode->totalAttr;
    res->tupleSize = jNode->tupleSize;
//    printf("res->totalAttr: %d\n", res->totalAttr);
//    printf("res->tupleSize: %d\n", res->tupleSize);
    res->attrType = (int *) malloc(res->totalAttr * sizeof(int));
    CHECK_POINTER(res->attrType);
    res->attrSize = (int *) malloc(res->totalAttr * sizeof(int));
    CHECK_POINTER(res->attrSize);
    res->attrIndex = (int *) malloc(res->totalAttr * sizeof(int));
    CHECK_POINTER(res->attrIndex);
    res->attrTotalSize = (int *) malloc(res->totalAttr * sizeof(int));
    CHECK_POINTER(res->attrTotalSize);
    res->dataPos = (int *) malloc(res->totalAttr * sizeof(int));
    CHECK_POINTER(res->dataPos);
    res->dataFormat = (int *) malloc(res->totalAttr * sizeof(int));
    CHECK_POINTER(res->dataFormat);
    res->content = (char **) malloc(res->totalAttr * sizeof(char *));
    CHECK_POINTER(res->content);


//    printf("leftOutputAttrNum: %d\n", jNode->leftOutputAttrNum);
    for(int i=0;i<jNode->leftOutputAttrNum;i++)
    {
        int pos = jNode->leftPos[i];
        res->attrType[pos] = jNode->leftOutputAttrType[i];
        int index = jNode->leftOutputIndex[i];
        res->attrSize[pos] = jNode->leftTable->attrSize[index];
        res->dataFormat[pos] = UNCOMPRESSED;
    }

    for(int i=0;i<jNode->rightOutputAttrNum;i++)
    {
        int pos = jNode->rightPos[i];
            res->attrType[pos] = jNode->rightOutputAttrType[i];
            int index = jNode->rightOutputIndex[i];
            res->attrSize[pos] = jNode->rightTable->attrSize[index];
            res->dataFormat[pos] = UNCOMPRESSED;
    }

    int maxLeftJoin = 0, maxRightJoin = 0;
    maxLeftJoin = getMaxVal(jNode->leftTable->content[jNode->leftKeyIndex],
                            leftTupleNum,
                            jNode->leftTable->attrType[jNode->leftKeyIndex]);
    
    maxRightJoin = getMaxVal(jNode->rightTable->content[jNode->rightKeyIndex],
                            rightTupleNum,
                            jNode->rightTable->attrType[jNode->rightKeyIndex]);

    // scan to find uniq_k -- assume already known in DB, won't time this part
    int uniq_K = max(maxLeftJoin, maxRightJoin)+1;
//    printf("MATRIX_K: %d\n", uniq_K);
    /*
    int minLeft = 0, minRight = 0;
    minLeft = getMinVal(jNode->leftTable->content[jNode->leftKeyIndex],
                            leftTupleNum,
                            jNode->leftTable->attrType[jNode->leftKeyIndex]);

    minRight = getMinVal(jNode->rightTable->content[jNode->rightKeyIndex],
                            rightTupleNum,
                            jNode->rightTable->attrType[jNode->rightKeyIndex]);
    int shifted_K = min(minLeft, minRight);
    if (shifted_K > 100000) // quick hack to handle Date type as int
    {
        
    }
    else
    {
        MATRIX_K = uniq_K; // MATRIX_K to determine sparsity
    }
    */
    MATRIX_K = uniq_K; // MATRIX_K to determine sparsity
    //MATRIX_K = *matrix_dim; // TODO: remove this later
    MATRIX_M = leftTupleNum;
    MATRIX_N = rightTupleNum;

    // assume each row contains only 1 element, Xnnz == XtupleNum
    Annz = MATRIX_M;
    Bnnz = MATRIX_N;

    long foreignKeySize = jNode->leftTable->attrTotalSize[jNode->leftKeyIndex];
    long primaryKeySize = jNode->rightTable->attrTotalSize[jNode->rightKeyIndex];
    
    int gbConstant = 0;   // 0: has groupBy, 1: no groupBy keyword
    int gbLeftRight = -1; // 0: gb by left, 1: gb by right
    int gbMatWidth = 0;   // size of dom(gb_column.val)

    int *lValIndex, *rValIndex;
    int ldataColIndex = -1;
    int ldata2 = -1;
    //int *ldataColIndex;
    int rdataColIndex = -1;
    int lgbIndex = -1, rgbIndex = -1;

    int quantizedScale = 1;  // quantization scale
    //int quantizedScale = 7237036;
    //ldataColIndex = (int *)malloc(sizeof(int) * jNode->leftOutputAttrNum);
    //memset(ldataColIndex, -1, sizeof(int) * jNode->leftOutputAttrNum);

    if (gb && (gb->groupByColNum == 1 && gb->groupByIndex[0] == -1)) 
    {
        gbConstant = 1;
    }
        
    // update MATRIX_M or MATRIX_N given groupBy keyword
    // FIXME: pure matrix-multiplication result may be affected
    // handle func first or dense/sparse first -> then update gbMatWidth
    // may have to move to later section
    if (gb && gbConstant != 1) // contains groupBy keyword
    {
        char *gb_column;
        // linear scan to find the max value of groupBy column 
        gbLeftRight = getGbLeftRight(jNode, gb, gbConstant, gbLeftRight);
        if (gbLeftRight == 0) {
            gb_column = jNode->leftTable->content[gb->groupByIndex[0]];

            gbMatWidth = getMaxVal(gb_column, jNode->leftTable->tupleNum, jNode->leftOutputAttrType[0]) + 1;
#ifdef DEBUG
            printf("matA gbMatWidth: %d\n", gbMatWidth);
#endif
            MATRIX_M = gbMatWidth;
        } 
        else if (gbLeftRight == 1) 
        {
            gb_column = jNode->rightTable->content[gb->groupByIndex[0]];
            gbMatWidth = getMaxVal(gb_column, jNode->rightTable->tupleNum, jNode->rightOutputAttrType[0]) + 1;
#ifdef DEBUG
            printf("matB gbMatWidth: %d\n", gbMatWidth);
#endif
            MATRIX_N = gbMatWidth;
        } 
        else 
        {
            printf("No matched column found.\n");
        }

        //int *lValIndex, *rValIndex;
        //int dataColIndex = -1;
        //int lgbIndex = -1, rgbIndex = -1;
        lValIndex = (int *)malloc(sizeof(int) * jNode->leftOutputAttrNum);
        rValIndex = (int *)malloc(sizeof(int) * jNode->rightOutputAttrNum);
        memset(lValIndex, -1, sizeof(int) * jNode->leftOutputAttrNum);
        memset(rValIndex, -1, sizeof(int) * jNode->rightOutputAttrNum);

        getValIndex(jNode, gb, lValIndex, rValIndex, lgbIndex, rgbIndex, 
                ldataColIndex, rdataColIndex, ldata2);
#ifdef DEBUG
        
        printf("numFuncExpCol: %d\n", gb->numFuncExpCol);
        printf("lValIndex[0]: %d\n", lValIndex[0]);
        printf("rValIndex[0]: %d\n", rValIndex[0]);
        printf("lgbIndex: %d\n", lgbIndex);
        printf("rgbIndex: %d\n", rgbIndex);
        printf("ldataColIndex: %d\n", ldataColIndex);
        printf("rdataColIndex: %d\n", rdataColIndex);
        
#endif

    } // end of contains groupBy keyword

    //FIXME: hard code for p_brand1
    int update_M = getMaxVal(jNode->leftTable->content[ldata2],jNode->leftTable->tupleNum, 4);
//    printf("p_brand1 max: %d\n", update_M+1);
    MATRIX_M = update_M+1;

    quantizedScale = getMaxVal(jNode->leftTable->content[ldataColIndex],
            jNode->leftTable->tupleNum, jNode->leftOutputAttrType[0]);
//    printf("quantizedScale: %d\n", quantizedScale);

    // parse data value index from gbNode
    //if (gb) {
    //    getValIndex(jNode, gb, lValIndex, rValIndex, lgbIndex, rgbIndex, dataColIndex);
    //}

    //#ifdef DEBUG
        //cudaPrintfInit();
    //#endif
    //    clock_gettime(CLOCK_REALTIME, &tcu_start);
    //    clock_gettime(CLOCK_REALTIME, &init_start);
    struct gpu_timer initStart, end2endStart;
    // read row data from column store
    char *gpu_fact, *gpu_dim;         // joined column index
    char *gpu_ldata, *gpu_rdata;      // data columns of left/right tables
    char *d_redMat;
    half *d_redMatFp16;
    char *gpu_ldata2;

    float alpha = 1.0f;
    float beta = 0.0f;
    half *d_fp16_A, *d_fp16_BT;
    float *c_cublas;
    half *c_fp16_cublas;

    half alpha_fp16 = __float2half(1.0f);
    half beta_fp16 = __float2half(1.0f);
    float *c_host_cublas;

//    struct timespec gbCount_start, gbCount_end;

    hipblasHandle_t cublasHandle;
    hipEvent_t startcublasEX;
    hipEvent_t stopcublasEX;

    cudaErrCheck(hipEventCreate(&startcublasEX));
    cudaErrCheck(hipEventCreate(&stopcublasEX));
    cublasErrCheck(hipblasCreate(&cublasHandle));
    cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH));
    //cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH));


    // allocate device memory for inputs
        
    //long foreignKeySize = jNode->leftTable->attrTotalSize[jNode->leftKeyIndex];
    //long primaryKeySize = jNode->rightTable->attrTotalSize[jNode->rightKeyIndex];

    //printf("gpu_fact size: %d\tgpu_dim size: %d\n", foreignKeySize, primaryKeySize);
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpu_fact,foreignKeySize));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpu_dim,primaryKeySize));


    // groupBy on value other than join key
    if (gb && gbConstant != 1) 
    {

        if (lValIndex[0] != -1 || lgbIndex != -1) 
        {
            CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpu_ldata,foreignKeySize));
            CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpu_ldata2,foreignKeySize));
#ifdef DEBUG
            printf("hipMalloc left_data column\n");
#endif
        }

        if (rValIndex[0] != -1 || rgbIndex != -1) {
            CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&gpu_rdata,primaryKeySize));
#ifdef DEBUG
            printf("hipMalloc right_data column\n");
#endif
        }
    }


    c_host_cublas = (float*)calloc(MATRIX_M*MATRIX_N, sizeof(float));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&c_cublas,(uint64_t)MATRIX_M*(uint64_t)MATRIX_N*sizeof(float)));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&d_fp16_A,(uint64_t)MATRIX_M*(uint64_t)MATRIX_K*sizeof(half)));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&d_fp16_BT,(uint64_t)MATRIX_K*(uint64_t)MATRIX_N*sizeof(half)));

    initTime = initStart.milliseconds_elapsed();
    //clock_gettime(CLOCK_REALTIME, &init_end);
    
    //clock_gettime(CLOCK_REALTIME, &cuMemcpy_start);
    struct gpu_timer cudaMemcpyStart;
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpu_fact,jNode->leftTable->content[jNode->leftKeyIndex], foreignKeySize,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpu_dim,jNode->rightTable->content[jNode->rightKeyIndex], primaryKeySize,hipMemcpyHostToDevice));

    if (gb && gbConstant != 1) 
    {
        if (lValIndex[0] != -1 || lgbIndex != -1) 
        {
            //TODO: how to copy all OutputAttr
            CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpu_ldata,jNode->leftTable->content[ldataColIndex], foreignKeySize,hipMemcpyHostToDevice));
#ifdef DEBUG
            printf("hipMemcpy gpu_ldata\n");
#endif
        }

        // only for q2_1 test
        if (ldata2 != -1) {
            CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpu_ldata2,jNode->leftTable->content[ldata2], foreignKeySize,hipMemcpyHostToDevice));
#ifdef DEBUG
            printf("hipMemcpy gpu_ldata2\n");
#endif
        }

        if (rValIndex[0] != -1 || rgbIndex != -1) 
        {
            CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(gpu_rdata,jNode->rightTable->content[rdataColIndex], primaryKeySize,hipMemcpyHostToDevice));
#ifdef DEBUG
            printf("hipMemcpy gpu_rdata\n");
#endif
        }
    }

    cudaMemcpyTime = cudaMemcpyStart.milliseconds_elapsed();
    //clock_gettime(CLOCK_REALTIME, &cuMemcpy_end);

    //clock_gettime(CLOCK_REALTIME, &fill_start);

    //TODO: determine whether to use normal filling method or cuSPARSE filling
    // assume sparsity, A_nnz, B_nnz is given (scan from host function) 
    // if sparsity > delta && A_nnz && B_nnz
        if (gb && gb->gbExp[gb->aggFuncIndex].func == SUM) 
        {

            if (gb->numFuncExpCol == 1) 
            {
                if (rValIndex[0] == -1)
                {
                    //clock_gettime(CLOCK_REALTIME, &fill_start);
                    struct gpu_timer fillStart;

                    if (ldata2 != -1) {
                        //printf("calling gpu_fill_2data\n");
                        gpu_fill_2data<<<(MAX_THREADS+leftTupleNum-1)/MAX_THREADS,MAX_THREADS>>> (gpu_fact,
                                gpu_ldata,
                                gpu_ldata2,
                                MATRIX_K,
                                d_fp16_A,
                                leftTupleNum,
                                jNode->leftTable->attrType[jNode->leftKeyIndex],
                                quantizedScale);
                        // update MATIRX_M 

                    } else {
                        //printf("calling gpu_fill_data\n");
                        gpu_fill_data<<<(MAX_THREADS+leftTupleNum-1)/MAX_THREADS,MAX_THREADS>>> (gpu_fact,
                        gpu_ldata,    
                        MATRIX_K,
                        d_fp16_A,
                        leftTupleNum,
                        jNode->leftTable->attrType[jNode->leftKeyIndex]);

                    }
                    cudaErrCheck(hipFree(gpu_fact));
                    cudaErrCheck(hipFree(gpu_ldata));
                
                    gpu_fill_gb_transpose<<<(MAX_THREADS+rightTupleNum-1)/MAX_THREADS,MAX_THREADS>>> (
                        gpu_dim,
                        gpu_rdata,
                        MATRIX_N,
                        d_fp16_BT,
                        rightTupleNum,
                        jNode->rightTable->attrType[jNode->rightKeyIndex]);
                    fillTime = fillStart.milliseconds_elapsed();
                    //clock_gettime(CLOCK_REALTIME, &fill_end); 
    
                    cudaErrCheck(hipFree(gpu_dim));
                    cudaErrCheck(hipFree(gpu_rdata));
    
                    printf("MxK:%dx%d\tKxN:%dx%d\n", MATRIX_M, MATRIX_K, MATRIX_K, MATRIX_N);
                    printf("Running GemmEx (Group-by aggregates) on TCUs...\n");
                    cudaErrCheck(hipEventRecord(startcublasEX));
                    struct gpu_timer compute_start; 
                    cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                            MATRIX_N, MATRIX_M, MATRIX_K,
                            &alpha,
                            d_fp16_BT, HIP_R_16F, MATRIX_N,
                            d_fp16_A, HIP_R_16F, MATRIX_K,
                            &beta,
                            c_cublas, HIP_R_32F, MATRIX_N,
                            HIPBLAS_COMPUTE_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP));
                    cudaErrCheck(hipEventRecord(stopcublasEX));
                    tcu_compute_time = compute_start.milliseconds_elapsed();
                    cudaErrCheck(hipFree(d_fp16_A));
                    cudaErrCheck(hipFree(d_fp16_BT));

                    /*
                    float *check = (float*)malloc(MATRIX_M * MATRIX_N * sizeof(float));
                    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(check, c_cublas, MATRIX_M * MATRIX_N * sizeof(float),hipMemcpyDeviceToHost));
                    for (int i = 0; i < MATRIX_M*MATRIX_N; i++) {
                        if (check[i] != 0.0f) {
                        //if (check[i] > 0.0000001) {
                            //printf("%f\n", check[i]);
                            printf("%f\n", check[i] * quantizedScale);
                        }
                    }*/


                // TODO: if sparsity over threshold -- calculate join count using cusparse<t>nnz()
//                float *test = (float*)malloc(sizeof(float) * MATRIX_M * MATRIX_N);

//                int *d_jCount, *h_jCount;
//                CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&d_jCount, (MATRIX_M*MATRIX_N+255)/256 * sizeof(int)));
//                CUDA_SAFE_CALL_NO_SYNC(hipMemset(d_jCount, 0, (MATRIX_M*MATRIX_N+255)/256 * sizeof(int)));
//                h_jCount = (int*)malloc((MATRIX_M*MATRIX_N+255)/256 * sizeof(int));
                //naiveCount<<<(MATRIX_M*MATRIX_N+255)/256, 256>>> (c_cublas, MATRIX_M*MATRIX_N, d_jCount);

                

                    //reductionCount<<<(MATRIX_M*MATRIX_N+255)/256, 256>>> (d_jCount, c_cublas, MATRIX_M*MATRIX_N);
                    // TODO: if gbConstant != 1 then need to copy c_cublas first before placeCount<<<>>> 
                    // otherwise, the in-place replacement will affect the following groupBy operation
                    placeCount<<<(MATRIX_M*MATRIX_N+255)/256, 256>>> (c_cublas, c_cublas, MATRIX_M*MATRIX_N);
                    printf("Join Count: %lld\n", getCount(c_cublas, MATRIX_M*MATRIX_N));


                    /*
                    cudaErrCheck(hipMemcpy(h_jCount, d_jCount, (MATRIX_M*MATRIX_N+255)/256 * sizeof(int), hipMemcpyDeviceToHost));
                    int tmpCount = 0;
                    for (int i = 0; i <(MATRIX_M*MATRIX_N+255)/256; i++) 
                    {
                        tmpCount += h_jCount[i];
                        printf("i: %d\th_jCount: \n", i, h_jCount[i]);
                    }
                    printf("Join Count: %d\n", tmpCount);
                    */
    
                    //printf("Join Count: %d\n", h_jCount[0]);
                    /*
                    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(test, c_cublas, 
                                           MATRIX_M * MATRIX_N, 
                                           hipMemcpyDeviceToHost));
                    print_matrix(test, MATRIX_M, MATRIX_N);*/
                
                    if (gbConstant != 1) 
                    {
                        struct gpu_timer groupBy_start;
                        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&d_redMat, 1 * MATRIX_M * sizeof(char)));
                        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&d_redMatFp16, 1 * MATRIX_M * sizeof(half)));
                        CUDA_SAFE_CALL_NO_SYNC(hipMemset(d_redMat, 1, MATRIX_M * sizeof(char)));
                        convertCharToFp16 <<< (MATRIX_M + 255) / 256, 256 >>> (d_redMatFp16, 
                            d_redMat, MATRIX_M);
                        cudaErrCheck(hipFree(d_redMat));
                    
                        // compute groupBy count by performing reduction
                        half *temp_c;
                        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&temp_c, MATRIX_M * MATRIX_N * sizeof(half)));
                        
                        convertFp32ToFp16 <<< (MATRIX_M * MATRIX_N + 255) / 256, 256 >>> (temp_c, c_cublas, MATRIX_M * MATRIX_N);
                        cudaErrCheck(hipFree(c_cublas));
            
                        float *d_reduction_res;
                        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&d_reduction_res, 1 * MATRIX_N * sizeof(float)));
            
                        printf("Perform groupBy reduction...\n");
                        cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                            MATRIX_N, 1, MATRIX_M,
                            &alpha,
                            temp_c, HIP_R_16F, MATRIX_N,
                            d_redMatFp16, HIP_R_16F, MATRIX_M,
                            &beta,
                            d_reduction_res, HIP_R_32F, MATRIX_N,
                            HIPBLAS_COMPUTE_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP));
            
                        cudaErrCheck(hipFree(temp_c));
                        cudaErrCheck(hipFree(d_redMatFp16));
            
                        int *d_gbCount;
                        int h_gbCount = 0;
                        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&d_gbCount, 1 * sizeof(int)));
                        CUDA_SAFE_CALL_NO_SYNC(hipMemset(d_gbCount, 0, 1 * sizeof(int)));
                        
                        //groupByCount<<<(MATRIX_N+255), 256>>> (d_reduction_res, MATRIX_N, d_gbCount);
                        naiveCount<<<(MATRIX_N+255)/256, 256>>> (d_reduction_res, MATRIX_N, d_gbCount);
                        cudaErrCheck(hipFree(d_reduction_res));
                        cudaErrCheck(hipMemcpy(&h_gbCount, d_gbCount, 1 * sizeof(int), hipMemcpyDeviceToHost));
                        tcu_groupBy_time = groupBy_start.milliseconds_elapsed();
                        cudaErrCheck(hipFree(d_gbCount));
                        printf("GroupBy Count: %d\n", h_gbCount);
        
                    }
                } // end of rValIndex[0] == -1 
                else if (lValIndex[0] == -1) 
                {
                    //clock_gettime(CLOCK_REALTIME, &fill_start);
                    struct gpu_timer fillStart;
                    gpu_fill_gb<<<(MAX_THREADS+leftTupleNum-1)/MAX_THREADS,MAX_THREADS>>> (
                        gpu_fact,
                        gpu_ldata,    
                        MATRIX_K,
                        d_fp16_A,
                        leftTupleNum,
                        jNode->leftTable->attrType[jNode->leftKeyIndex]);
                    cudaErrCheck(hipFree(gpu_fact));
                    cudaErrCheck(hipFree(gpu_ldata));
    
                    gpu_fill_data_transpose<<<(MAX_THREADS+rightTupleNum-1)/MAX_THREADS,MAX_THREADS>>> (
                        gpu_dim,
                        gpu_rdata,
                        MATRIX_N,
                        d_fp16_BT,
                        rightTupleNum,
                        jNode->rightTable->attrType[jNode->rightKeyIndex]);
                    fillTime = fillStart.milliseconds_elapsed();
                    //clock_gettime(CLOCK_REALTIME, &fill_end);
                    cudaErrCheck(hipFree(gpu_dim));
                    cudaErrCheck(hipFree(gpu_rdata));
    
                    printf("MxK:%dx%d\tKxN:%dx%d\n", MATRIX_M, MATRIX_K, MATRIX_K, MATRIX_N);
                    printf("Running GemmEx (Group-by aggregates) on TCUs...\n");
                    cudaErrCheck(hipEventRecord(startcublasEX));
                    cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                        MATRIX_N, MATRIX_M, MATRIX_K,
                        &alpha,
                        d_fp16_BT, HIP_R_16F, MATRIX_N,
                        d_fp16_A, HIP_R_16F, MATRIX_K,
                        &beta,
                        c_cublas, HIP_R_32F, MATRIX_N,
                        HIPBLAS_COMPUTE_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP));
                    cudaErrCheck(hipEventRecord(stopcublasEX));
                    cudaErrCheck(hipFree(d_fp16_A));
                    cudaErrCheck(hipFree(d_fp16_BT));

                    if (gbConstant != 1) 
                    {
                        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&d_redMat, 1 * MATRIX_N * sizeof(char)));
                        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&d_redMatFp16, 1 * MATRIX_N * sizeof(half)));
                        CUDA_SAFE_CALL_NO_SYNC(hipMemset(d_redMat, 1, MATRIX_N * sizeof(char)));
                        convertCharToFp16 <<< (MATRIX_N + 255) / 256, 256 >>> (d_redMatFp16, 
                            d_redMat, MATRIX_N);
                        cudaErrCheck(hipFree(d_redMat));
                    
                        half *temp_c;
                        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&temp_c, MATRIX_M * MATRIX_N * sizeof(half)));
                        
                        convertFp32ToFp16 <<< (MATRIX_M * MATRIX_N + 255) / 256, 256 >>> (temp_c, c_cublas, MATRIX_M * MATRIX_N);
                        cudaErrCheck(hipFree(c_cublas));
            
                        float *d_reduction_res;
                        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&d_reduction_res, MATRIX_M * 1 * sizeof(float)));
            
                        printf("Perform groupBy reduction...\n");
                        cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                                1, MATRIX_M, MATRIX_N,
                                &alpha,
                                d_redMatFp16, HIP_R_16F, 1,
                                temp_c, HIP_R_16F, MATRIX_N,
                                &beta,
                                d_reduction_res, HIP_R_32F, 1,
                                HIPBLAS_COMPUTE_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP));
            
                        cudaErrCheck(hipFree(temp_c));
                        cudaErrCheck(hipFree(d_redMatFp16));
            
                        int *d_gbCount; 
                        int h_gbCount = 0;
                        CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void**)&d_gbCount, 1 * sizeof(int)));
                        CUDA_SAFE_CALL_NO_SYNC(hipMemset(d_gbCount, 0, 1 * sizeof(int)));
                        
                        //groupByCount<<<(MATRIX_M+255), 256>>> (d_reduction_res, MATRIX_M, d_gbCount);
                        naiveCount<<<(MATRIX_M+255)/256, 256>>> (d_reduction_res, MATRIX_M, d_gbCount);
                        cudaErrCheck(hipFree(d_reduction_res));
                        cudaErrCheck(hipMemcpy(&h_gbCount, d_gbCount, 1 * sizeof(int), hipMemcpyDeviceToHost));
                        cudaErrCheck(hipFree(d_gbCount));
                        printf("GroupBy Count: %d\n", h_gbCount);
                    }
                } // end of lValIndex[0] == -1
            } // end of gb->numFuncExpCol == 1
            else if (gb->numFuncExpCol == 2 && gb->math_op == MULTIPLY) 
            {
                //clock_gettime(CLOCK_REALTIME, &fill_start);
                struct gpu_timer fillStart;
                gpu_fill_data<<<(MAX_THREADS+leftTupleNum-1)/MAX_THREADS,MAX_THREADS>>> (gpu_fact,
                    gpu_ldata,    
                    MATRIX_K,
                    d_fp16_A,
                    leftTupleNum,
                    jNode->leftTable->attrType[jNode->leftKeyIndex]);
    
                cudaErrCheck(hipFree(gpu_fact));
                cudaErrCheck(hipFree(gpu_ldata));
    
                gpu_fill_data_transpose<<<(MAX_THREADS+rightTupleNum-1)/MAX_THREADS,MAX_THREADS>>> (
                    gpu_dim,
                    gpu_rdata,
                    MATRIX_N,
                    d_fp16_BT,
                    rightTupleNum,
                    jNode->rightTable->attrType[jNode->rightKeyIndex]);
                fillTime = fillStart.milliseconds_elapsed();
                //clock_gettime(CLOCK_REALTIME, &fill_end);
                cudaErrCheck(hipFree(gpu_dim));
                cudaErrCheck(hipFree(gpu_rdata));
    
                printf("MxK:%dx%d\tKxN:%dx%d\n", MATRIX_M, MATRIX_K, MATRIX_K, MATRIX_N);
                printf("Running GemmEx (Aggregation) on TCUs...\n");
                cudaErrCheck(hipEventRecord(startcublasEX));
                cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                    MATRIX_N, MATRIX_M, MATRIX_K,
                    &alpha,
                    d_fp16_BT, HIP_R_16F, MATRIX_N,
                    d_fp16_A, HIP_R_16F, MATRIX_K,
                    &beta,
                    c_cublas, HIP_R_32F, MATRIX_N,
                    HIPBLAS_COMPUTE_32F, CUBLAS_GEMM_DEFAULT_TENSOR_OP));
                cudaErrCheck(hipEventRecord(stopcublasEX));
                cudaErrCheck(hipFree(d_fp16_A));
                cudaErrCheck(hipFree(d_fp16_BT));
    
                //float cublasEXTime;
                //cudaErrCheck(hipEventSynchronize(stopcublasEX));
                //cudaErrCheck(hipEventElapsedTime(&cublasEXTime, startcublasEX, stopcublasEX));
                //printf("cublasEX tensor cores (FP16) took %fms\n", cublasEXTime);

#ifdef DEBUG
                float *testRes;
                testRes = (float*)malloc(MATRIX_M * MATRIX_N * sizeof(float));
                cudaErrCheck(hipMemcpy(testRes, c_cublas, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyDeviceToHost));
    
                print_matrix(testRes, MATRIX_M, MATRIX_N);
#endif
            } // end of SUM(MULTIPLY 2 data values)

        } // end of func == SUM 
        else if (gb->gbExp[gb->aggFuncIndex].func == COUNT) 
        {
            //clock_gettime(CLOCK_REALTIME, &fill_start);
            // TODO: filling method for COUNT?
            //clock_gettime(CLOCK_REALTIME, &fill_end);
        } 
        else // simply return join_count (general MM)
        {
            //clock_gettime(CLOCK_REALTIME, &fill_start);

            //clock_gettime(CLOCK_REALTIME, &fill_end);
        }
    
//    clock_gettime(CLOCK_REALTIME, &fill_end); 

    // free those data structures
    end2endTime = end2endStart.milliseconds_elapsed();
//    clock_gettime(CLOCK_REALTIME, &tcu_end);


    float cublasEXTime;
    cudaErrCheck(hipEventSynchronize(stopcublasEX));
    cudaErrCheck(hipEventElapsedTime(&cublasEXTime, startcublasEX, stopcublasEX));

    //double init_elapse = (init_end.tv_sec -  init_start.tv_sec)* BILLION + init_end.tv_nsec - init_start.tv_nsec;
    //double cuMemcpy_elapse = (cuMemcpy_end.tv_sec -  cuMemcpy_start.tv_sec)* BILLION + cuMemcpy_end.tv_nsec - cuMemcpy_start.tv_nsec;
    //double tcu_fill = (fill_end.tv_sec -  fill_start.tv_sec)* BILLION + fill_end.tv_nsec - fill_start.tv_nsec;
    //double tcu_elapse = (tcu_end.tv_sec -  tcu_start.tv_sec)* BILLION + tcu_end.tv_nsec - tcu_start.tv_nsec;
//    double count_elapse = (count_end.tv_sec -  count_start.tv_sec)* BILLION + count_end.tv_nsec - count_start.tv_nsec;

    /*
    printf("Initialization: %lf(ms)\n", init_elapse/(1000*1000));
    printf("gpu_timer -- init time: %fms\n\n", initTime);
    printf("hipMemcpy: %lf(ms)\n", cuMemcpy_elapse/(1000*1000));
    printf("gpu_timer -- hipMemcpy time: %fms\n\n", cudaMemcpyTime);
    printf("Matrices filling: %lf(ms)\n", tcu_fill/(1000*1000));
    printf("gpu_timer -- fill time: %fms\n\n", fillTime);
    printf("cublasEX tensor cores (FP16) took %fms\n", cublasEXTime);
    printf("gpu_timer -- tcu compute time: %fms\n", tcu_compute_time);
    printf("gpu_timer -- tcu groupBy time: %fms\n\n", tcu_groupBy_time);
    printf("MMA total time: %lf(ms)\n", tcu_elapse/(1000*1000));
    printf("gpu_timer -- end-to-end time: %fms\n\n", end2endTime);
    */
    printf("Initialization:   %f ms\n"  , initTime);
    printf("hipMemcpy:       %f ms\n"      , cudaMemcpyTime);
    printf("Matrices filling: %f ms\n", fillTime);
    printf("TCU compute time: %f ms\n"  , tcu_compute_time);
    printf("TCU groupBy time: %f ms\n"  , tcu_groupBy_time);
    printf("End-to-end time:  %f ms\n"   , end2endTime);

//if (gb && (gb->gbExp[1].func == COUNT || gb->gbExp[0].func == COUNT)) {
//    printf("cublasEX sum counting: %lf(ms)\n", count_elapse/(1000*1000));
//}

//#ifdef DEBUG
    //cudaPrintfDisplay(stdout, true);
    //cudaPrintfEnd();
//#endif
    return res; // FIXME: return res table if second join need this as input  

}
